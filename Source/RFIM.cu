/*
 * RFIM.cu
 *
 *  Created on: 17/03/2016
 *      Author: vincentvillani
 */

#include "../Header/RFIM.h"


#include "../Header/UtilityFunctions.h"
#include "../Header/RFIMHelperFunctions.h"

#include <stdio.h>

void RFIMRoutine(RFIMMemoryStruct* RFIMStruct, float* d_columnMajorSignalMatrices, float* d_columnMajorFilteredSignalMatrices)
{


	//If we reduce everything, we will have nothing left...
	if(RFIMStruct->h_eigenVectorDimensionsToReduce >= RFIMStruct->h_valuesPerSample)
	{
		fprintf(stderr, "RFIMStruct->h_eigenVectorDimensionsToReduce >= RFIMStruct->h_valuesPerSample\n");
		exit(1);
	}

	//Calculate covariance matrix for this signal
	Device_CalculateCovarianceMatrix(RFIMStruct, d_columnMajorSignalMatrices);



	//Calculate the eigenvectors/values
	Device_EigenvalueSolver(RFIMStruct);



	//Project the signal against the reduced eigenvector matrix and back again to the original dimensions
	Device_EigenReductionAndFiltering(RFIMStruct, d_columnMajorSignalMatrices, d_columnMajorFilteredSignalMatrices);


	//Make sure all streams we used are done computing before we leave here
	//This is done to ensure some when these streams are used again, we don't override memory other streams may need
	//(some streams may overtake others and be working in a whole different RFIMRoutine iteration and overwrite needed memory)
	hipError_t hipError_t;
	for(uint64_t i = 0; i < RFIMStruct->h_cudaStreamsLength; ++i)
	{
		hipError_t = hipStreamSynchronize(RFIMStruct->h_cudaStreams[i]);

		if(hipError_t != hipSuccess)
		{
			fprintf(stderr, "RFIMRoutine: Something went wrong along the way...\n");
			exit(1);
		}
	}

}



void RFIMRoutineComplex(RFIMMemoryStructComplex* RFIMStruct, hipComplex* d_columnMajorSignalMatrices, hipComplex* d_columnMajorFilteredSignalMatrices)
{
	//If we reduce everything, we will have nothing left...
	if(RFIMStruct->h_eigenVectorDimensionsToReduce >= RFIMStruct->h_valuesPerSample)
	{
		fprintf(stderr, "RFIMStruct->h_eigenVectorDimensionsToReduce >= RFIMStruct->h_valuesPerSample\n");
		exit(1);
	}

	//Calculate covariance matrix for this signal
	Device_CalculateCovarianceMatrixComplex(RFIMStruct, d_columnMajorSignalMatrices);



	//Calculate the eigenvectors/values
	Device_EigenvalueSolverComplex(RFIMStruct);



	//Project the signal against the reduced eigenvector matrix and back again to the original dimensions
	Device_EigenReductionAndFilteringComplex(RFIMStruct, d_columnMajorSignalMatrices, d_columnMajorFilteredSignalMatrices);


	//Make sure all streams we used are done computing before we leave here
	//This is done to ensure some when these streams are used again, we don't override memory other streams may need
	//(some streams may overtake others and be working in a whole different RFIMRoutine iteration and overwrite needed memory)
	hipError_t hipError_t;
	for(uint64_t i = 0; i < RFIMStruct->h_cudaStreamsLength; ++i)
	{
		hipError_t = hipStreamSynchronize(RFIMStruct->h_cudaStreams[i]);

		if(hipError_t != hipSuccess)
		{
			fprintf(stderr, "RFIMRoutine: Something went wrong along the way...\n");
			exit(1);
		}
	}
}


