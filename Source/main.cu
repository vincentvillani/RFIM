
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include <stdint.h>

#include "../Header/Kernels.h"
#include "../Header/UnitTests.h"
#include "../Header/CudaMacros.h"
#include "../Header/RFIMHelperFunctions.h"



int main(int argc, char **argv)
{
	//Run all the unit tests
	RunAllUnitTests();


	//1. Generate a signal on the device
	//----------------------------------
	uint64_t h_valuesPerSample = 26;
	uint64_t h_numberOfSamples = 1024;

	float* d_whiteNoiseSignalMatrix = Device_GenerateWhiteNoiseSignal(h_valuesPerSample, h_numberOfSamples);

	//----------------------------------

	//2.Calculate the covariance matrix of this signal
	//----------------------------------
	float* d_covarianceMatrix = Device_CalculateCovarianceMatrix(d_whiteNoiseSignalMatrix, h_valuesPerSample, h_numberOfSamples);

	//----------------------------------

	//3. Graph the covariance matrix
	//----------------------------------
	//http://gnuplot.sourceforge.net/demo/heatmaps.html

	//----------------------------------


	//Free all memory
	//----------------------------------

	hipFree(d_whiteNoiseSignalMatrix);
	hipFree(d_covarianceMatrix);

	//----------------------------------

	return 0;
}
