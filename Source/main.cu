
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <string>

#include <stdint.h>

#include "../Header/Kernels.h"
#include "../Header/UnitTests.h"
#include "../Header/CudaMacros.h"
#include "../Header/RFIMHelperFunctions.h"
#include "../Header/CudaUtilityFunctions.h"
#include "../Header/UtilityFunctions.h"

//TODO: Make the user hand in a cublasHandle to use in the RFIMHelper functions



int main(int argc, char **argv)
{
	//Run all the unit tests
	RunAllUnitTests();


	//1. Generate a signal on the device
	//----------------------------------

	uint64_t h_valuesPerSample = 26;
	uint64_t h_numberOfSamples = 1024;

	//Start cuda rand library
	hiprandGenerator_t rngGen;

	if( hiprandCreateGenerator(&rngGen, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "main: Unable to start cuRand library\n");
		exit(1);
	}

	//Set the RNG seed
	if((hiprandSetPseudoRandomGeneratorSeed(rngGen, 1)) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "main: Unable to set the RNG Seed value\n");
		exit(1);
	}



	float* d_whiteNoiseSignalMatrix = Device_GenerateWhiteNoiseSignal(&rngGen, h_valuesPerSample, h_numberOfSamples);

	//Destroy the RNG
	if(hiprandDestroyGenerator(rngGen) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Error in destroying the RNG generator \n");
		exit(1);
	}

	//----------------------------------

	//2.Calculate the covariance matrix of this signal
	//----------------------------------

	//Setup the cublas library
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	float* d_covarianceMatrix = Device_CalculateCovarianceMatrix(&cublasHandle, d_whiteNoiseSignalMatrix, h_valuesPerSample, h_numberOfSamples);


	//Destroy the cublas handle
	hipblasDestroy(cublasHandle);

	//----------------------------------

	//3. Graph the covariance matrix
	//----------------------------------

	//Transpose it to row-major (simplify writing to file)
	float* d_covarianceMatrixTranspose = Device_MatrixTranspose(d_covarianceMatrix, h_valuesPerSample, h_valuesPerSample);

	//Copy the signal to host memory
	float* h_covarianceMatrixTranspose = CudaUtility_CopySignalToHost(d_covarianceMatrixTranspose,
			h_valuesPerSample * h_valuesPerSample * sizeof(float));

	//Write the signal to file
	Utility_WriteSignalMatrixToFile(std::string("signal.txt"), h_covarianceMatrixTranspose, h_valuesPerSample, h_valuesPerSample);

	//Graph it via python on own computer!

	//----------------------------------


	//Free all memory
	//----------------------------------

	free(h_covarianceMatrixTranspose);

	hipFree(d_whiteNoiseSignalMatrix);
	hipFree(d_covarianceMatrix);
	hipFree(d_covarianceMatrixTranspose);

	//----------------------------------

	return 0;
}
