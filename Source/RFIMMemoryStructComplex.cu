/*
 * RFIMMemoryStructComplex.cu
 *
 *  Created on: 11 Apr 2016
 *      Author: vincentvillani
 */

#include <stdio.h>

#include "../Header/RFIMMemoryStructComplex.h"


RFIMMemoryStructComplex* RFIMMemoryStructComplexCreate(uint64_t h_valuesPerSample, uint64_t h_numberOfSamples, uint64_t h_dimensionToReduce,
		uint64_t h_batchSize, uint64_t h_numberOfCUDAStreams)
{


	RFIMMemoryStructComplex* result;
	hipHostMalloc(&result, sizeof(RFIMMemoryStructComplex));


	//Set signal attributes
	//------------------------
	result->h_valuesPerSample = h_valuesPerSample;
	result->h_numberOfSamples = h_numberOfSamples;
	result->h_eigenVectorDimensionsToReduce = h_dimensionToReduce;
	result->h_batchSize = h_batchSize;
	result->h_cudaStreamsLength = h_numberOfCUDAStreams;



	//Setup library handles
	//------------------------
	hipHostMalloc(&(result->cublasHandle), sizeof(hipblasHandle_t));
	hipHostMalloc(&(result->cusolverHandle), sizeof(hipsolverHandle_t));

	hipblasStatus_t hipblasStatus_t;
	hipsolverStatus_t cusolverStatus;

	//Create the contexts for each library
	hipblasStatus_t = hipblasCreate( result->cublasHandle );
	cusolverStatus = hipsolverDnCreate( result->cusolverHandle );

	//Check the contexts started up ok
	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cublas context\n");
		exit(1);
	}

	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cusolver context\n");
		exit(1);
	}


	//Allocate space for the cudaSteams
	hipHostMalloc(&(result->h_cudaStreams), sizeof(hipStream_t) * h_numberOfCUDAStreams);
	for(uint64_t i = 0; i < h_numberOfCUDAStreams; ++i)
	{
		hipStreamCreate(result->h_cudaStreams + i);
	}





	//Setup the one vec, we use the same memory over and over again, it should never change
	//------------------------
	uint64_t oneVecLength = h_numberOfSamples;
	uint64_t oneVecByteSize = sizeof(hipComplex) * oneVecLength;


	hipComplex* h_oneVec;
	hipHostMalloc(&h_oneVec, oneVecByteSize);
	hipMalloc(&(result->d_oneVec), oneVecByteSize);

	//Fill the one vec with ones
	for(uint64_t i = 0; i < oneVecLength; ++i)
	{
		h_oneVec[i] = make_hipComplex(1, 0);
	}

	//copy the ones over and free the host memory
	hipMemcpy(result->d_oneVec, h_oneVec, oneVecByteSize, hipMemcpyHostToDevice);
	hipHostFree(h_oneVec);





	//Setup the mean vec
	//------------------------
	uint64_t meanVecLength = h_valuesPerSample * h_batchSize;
	uint64_t meanVecByteSize = sizeof(hipComplex) * meanVecLength;

	result->h_meanVecBatchOffset = h_valuesPerSample;

	hipMalloc(&(result->d_meanVec), meanVecByteSize);





	//Setup the covariance matrix
	//------------------------
	uint64_t covarianceMatrixLength = h_valuesPerSample * h_valuesPerSample * h_batchSize;
	uint64_t covarianceMatrixByteSize = sizeof(hipComplex) * covarianceMatrixLength;

	result->h_covarianceMatrixBatchOffset = h_valuesPerSample * h_valuesPerSample;

	hipMalloc(&(result->d_covarianceMatrix), covarianceMatrixByteSize);




	//Setup the eigenvector/value variables
	//------------------------

	//U and VT
	uint64_t singleULength = h_valuesPerSample * h_valuesPerSample;
	uint64_t ULength = singleULength * h_batchSize;
	uint64_t UByteSize = sizeof(hipComplex) * ULength;

	hipMalloc(&(result->d_U), UByteSize);
	hipMalloc(&(result->d_VT), UByteSize); //VT is the same size as U

	result->h_UBatchOffset = singleULength;
	result->h_VTBatchOffset = singleULength;


	//S
	uint64_t singleSLength = h_valuesPerSample;
	uint64_t SLength = h_valuesPerSample * h_batchSize;
	uint64_t SByteLength = sizeof(hipComplex) * SLength;

	hipMalloc(&(result->d_S), SByteLength);

	result->h_SBatchOffset = singleSLength;


	//Eigenworking space length
	result->h_singleEigWorkingSpaceByteSize = 0;

	cusolverStatus = hipsolverDnSgesvd_bufferSize(*result->cusolverHandle, h_valuesPerSample, h_valuesPerSample, &(result->h_singleEigWorkingSpaceByteSize));

	//Check if it went well
	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error finding eigenvalue working buffer size\n");
		exit(1);
	}


	uint64_t totalEigenvalueWorkingspace = result->h_singleEigWorkingSpaceByteSize * h_batchSize;

	//Allocate memory for the eigen working space
	hipMalloc(&(result->d_eigenWorkingSpace), totalEigenvalueWorkingspace);

	result->h_eigenWorkingSpaceBatchOffset = result->h_singleEigWorkingSpaceByteSize / sizeof(int);


	//DevInfo
	uint64_t devInfoSingleLength = 1;
	uint64_t devInfoLength = devInfoSingleLength * h_batchSize;
	uint64_t devInfoByteSize = sizeof(int) * devInfoLength;

	hipMalloc(&(result->d_devInfo), devInfoByteSize);
	hipHostMalloc(&(result->h_devInfo), devInfoByteSize);

	result->h_devInfoBatchOffset = devInfoSingleLength;






	//Projected signal
	//------------------------
	uint64_t projectedSignalSingleLength = h_valuesPerSample * h_numberOfSamples;
	uint64_t projectedSignalLength = projectedSignalSingleLength * h_batchSize;
	uint64_t projectedSignalByteSize = sizeof(hipComplex) * projectedSignalLength;

	hipMalloc(&(result->d_projectedSignalMatrix), projectedSignalByteSize);

	result->h_projectedSignalBatchOffset = projectedSignalSingleLength;



	hipError_t error = hipDeviceSynchronize();

	if(error != hipSuccess)
	{
		fprintf(stderr, "RFIMMemoryStructCreate: An error occurred, we probably ran out of device memory...\n");
		exit(1);
	}


	return result;
}

void RFIMMemoryStructComplexDestroy(RFIMMemoryStructComplex* RFIMStruct)
{
	//Free device memory
	hipFree(RFIMStruct->d_oneVec);
	hipFree(RFIMStruct->d_meanVec);
	hipFree(RFIMStruct->d_covarianceMatrix);

	hipFree(RFIMStruct->d_U);
	hipFree(RFIMStruct->d_S);
	hipFree(RFIMStruct->d_VT);
	hipFree(RFIMStruct->d_eigenWorkingSpace);
	hipFree(RFIMStruct->d_devInfo);

	hipFree(RFIMStruct->d_projectedSignalMatrix);


	//Free host memory
	hipHostFree(RFIMStruct->h_devInfo);



	//Destroy the cuda library contexts
	hipblasDestroy(*RFIMStruct->cublasHandle);
	hipsolverDnDestroy(*RFIMStruct->cusolverHandle);


	hipHostFree(RFIMStruct->cublasHandle);
	hipHostFree(RFIMStruct->cusolverHandle);

	//Destroy the cuda streams
	for(uint64_t i = 0; i < RFIMStruct->h_cudaStreamsLength; ++i)
	{
		hipStreamDestroy(RFIMStruct->h_cudaStreams[i]);
	}
	hipHostFree(RFIMStruct->h_cudaStreams);

	//Deallocate the struct memory on the host
	hipHostFree(RFIMStruct);
}
