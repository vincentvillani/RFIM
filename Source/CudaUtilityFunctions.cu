/*
 * CudaUtilityFunction.cpp
 *
 *  Created on: 26 Apr 2016
 *      Author: vincentvillani
 */

#include "../Header/CudaUtilityFunctions.h"

#include <hip/hip_runtime.h>


float** CudaUtility_createBatchedDevicePointers(float* d_basePointer, uint64_t h_offset, uint64_t h_length)
{

	uint64_t resultBatchedPointersByteSize = sizeof(float*) * h_length;

	float** h_resultBatchedPointers;
	hipHostMalloc(&h_resultBatchedPointers, resultBatchedPointersByteSize);

	for(uint64_t i = 0; i < h_length; ++i)
	{
		h_resultBatchedPointers[i] = d_basePointer + (i * h_offset);
	}

	//Copy the pointers to the device
	float** d_resultBatchedPointers;
	hipMalloc(&d_resultBatchedPointers, resultBatchedPointersByteSize);
	hipMemcpy(d_resultBatchedPointers, h_resultBatchedPointers, resultBatchedPointersByteSize, hipMemcpyHostToDevice);

	//Free the host pointers
	hipHostFree(h_resultBatchedPointers);

	return d_resultBatchedPointers;
}
