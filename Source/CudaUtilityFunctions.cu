/*
 * CudaUtilityFunctions.cu
 *
 *  Created on: 17/03/2016
 *      Author: vincentvillani
 */


#include "../Header/CudaUtilityFunctions.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

//Copies data from the host to the device and returns a device pointer
void CudaUtility_CopySignalToDevice(float* h_signal, float** d_destination, uint64_t signalByteSize, hipStream_t* cudaStream)
{
	hipError_t hipError_t;

	hipError_t = hipMemcpyAsync(*d_destination, h_signal, signalByteSize, hipMemcpyHostToDevice, *cudaStream);
	hipStreamSynchronize(*cudaStream);

	//copy data from host to device
	if( hipError_t != hipSuccess)
	{
		fprintf(stderr, "CopySignalToDevice: error copying memory to the device\n");
		fprintf(stderr, "Cuda error code: %s\n", hipGetErrorString(hipError_t));
		//exit(1);
	}

}

//Copies data from the device to the host and returns a host pointer
void CudaUtility_CopySignalToHost(float* d_signal, float** h_destination, uint64_t signalByteSize, hipStream_t* cudaStream)
{
	hipError_t hipError_t;

	hipError_t = hipMemcpyAsync(*h_destination, d_signal, signalByteSize, hipMemcpyDeviceToHost, *cudaStream);
	hipStreamSynchronize(*cudaStream);


	//copy data from host to device
	if( hipError_t != hipSuccess)
	{
		fprintf(stderr, "CopySignalToHost: error copying memory to the host\n");
		fprintf(stderr, "Cuda error code: %s\n", hipGetErrorString(hipError_t));
		//exit(1);
	}

}



float** CudaUtility_BatchAllocateHostArrays(uint32_t numberOfArrays, uint64_t arrayByteSize)
{
	float** h_resultHostPointers; //= (float**)malloc(sizeof(float*) * numberOfArrays);
	hipHostMalloc(&h_resultHostPointers, sizeof(float*) * numberOfArrays); //Allocate paged memory


	for(uint32_t i = 0; i < numberOfArrays; ++i)
	{
		//Allocate space for each array
		hipHostMalloc( &h_resultHostPointers[i], arrayByteSize); //= (float*)malloc(arrayByteSize);
	}

	return h_resultHostPointers;
}


void CudaUtility_BatchDeallocateHostArrays(float** h_arrays, uint32_t numberOfArrays)
{
	for(uint32_t i = 0; i < numberOfArrays; ++i)
	{
		//Free arrays of data
		hipHostFree(h_arrays[i]);
	}

	//Free pointers
	hipHostFree(h_arrays);
}


float** CudaUtility_BatchAllocateDeviceArrays(uint32_t numberOfArrays, uint64_t arrayByteSize, hipStream_t* cudaStream)
{
	//Allocate space for the pointers
	float** h_resultDevicePointers; //= (float**)malloc(sizeof(float*) * numberOfArrays);
	hipHostMalloc(&h_resultDevicePointers, sizeof(float*) * numberOfArrays);

	float** d_result;

	hipMalloc(&d_result, sizeof(float*) * numberOfArrays);


	//Allocate space for each pointer and copy it's address into h_resultDevicePointers + i
	for(uint32_t i = 0; i < numberOfArrays; ++i)
	{
		//Allocate each array's memory and store pointers to it on the host
		hipMalloc(&(h_resultDevicePointers[i]), arrayByteSize);
	}

	//Copy all the pointers into device memory
	hipMemcpyAsync(d_result, h_resultDevicePointers, sizeof(float*) * numberOfArrays, hipMemcpyHostToDevice, *cudaStream);

	//Wait for the memcpy to occur
	hipStreamSynchronize(*cudaStream);

	hipHostFree(h_resultDevicePointers);

	return d_result;
}



void CudaUtility_BatchDeallocateDeviceArrays(float** d_arrays, uint32_t numberOfArrays, hipStream_t* cudaStream)
{
	//Copy the pointers to the host
	float** h_arraysDevicePointers; // = (float**)malloc(sizeof(float*) * numberOfArrays);
	hipHostMalloc(&h_arraysDevicePointers, sizeof(float*) * numberOfArrays);

	hipMemcpyAsync(h_arraysDevicePointers, d_arrays, sizeof(float*) * numberOfArrays, hipMemcpyDeviceToHost, *cudaStream);
	hipStreamSynchronize(*cudaStream);

	for(uint32_t i = 0; i < numberOfArrays; ++i)
	{
		//Free each array
		hipFree(h_arraysDevicePointers[i]);
	}

	//Free the device pointers
	hipFree(d_arrays);

	//Free memory on the host
	hipHostFree(h_arraysDevicePointers);
}


void CudaUtility_BatchCopyArraysHostToDevice(float** d_arrays, float** h_arrays, uint32_t numberOfArrays, uint64_t arrayByteSize, hipStream_t* cudaStream)
{
	uint64_t pointersArrayByteSize = sizeof(float*) * numberOfArrays;

	//Copy the device pointers to the host
	float** h_devicePointers; //= (float**)malloc(pointersArrayByteSize);
	hipHostMalloc(&h_devicePointers, pointersArrayByteSize);

	hipMemcpyAsync(h_devicePointers, d_arrays, pointersArrayByteSize, hipMemcpyDeviceToHost, *cudaStream);
	hipStreamSynchronize(*cudaStream);

	for(uint32_t i = 0; i < numberOfArrays; ++i)
	{
		//Copy the actual data across to each pointer
		hipMemcpyAsync(h_devicePointers[i], h_arrays[i], arrayByteSize, hipMemcpyHostToDevice, *cudaStream);
	}

	hipStreamSynchronize(*cudaStream); //Shouldn't matter that we sync here

	hipHostFree(h_devicePointers);
}



void CudaUtility_BatchCopyArraysDeviceToHost(float** d_arrays, float** h_arrays, uint32_t numberOfArrays, uint64_t arrayByteSize, hipStream_t* cudaStream)
{
	uint64_t pointersArrayByteSize = sizeof(float*) * numberOfArrays;

	//Copy the device pointers to the host
	float** h_devicePointers; // = (float**)malloc(pointersArrayByteSize);
	hipHostMalloc(&h_devicePointers, pointersArrayByteSize);

	hipMemcpyAsync(h_devicePointers, d_arrays, pointersArrayByteSize, hipMemcpyDeviceToHost, *cudaStream);
	hipStreamSynchronize(*cudaStream);

	for(uint32_t i = 0; i < numberOfArrays; ++i)
	{
		//Copy the actual data across to each pointer
		hipMemcpyAsync(h_arrays[i], h_devicePointers[i], arrayByteSize, hipMemcpyDeviceToHost, *cudaStream);
	}

	hipStreamSynchronize(*cudaStream);

	hipHostFree(h_devicePointers);


}

