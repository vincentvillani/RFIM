/*
 * CudaUtilityFunctions.cu
 *
 *  Created on: 17/03/2016
 *      Author: vincentvillani
 */


#include "../Header/CudaUtilityFunctions.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

//Copies data from the host to the device and returns a device pointer
void CudaUtility_CopySignalToDevice(float* h_signal, float** d_destination, uint64_t signalByteSize)
{
	hipError_t hipError_t;

	hipError_t = hipMemcpy(*d_destination, h_signal, signalByteSize, hipMemcpyHostToDevice);

	//copy data from host to device
	if( hipError_t != hipSuccess)
	{
		fprintf(stderr, "CopySignalToDevice: error copying memory to the device\n");
		fprintf(stderr, "Cuda error code: %s\n", hipGetErrorString(hipError_t));
		//exit(1);
	}

}

//Copies data from the device to the host and returns a host pointer
void CudaUtility_CopySignalToHost(float* d_signal, float** h_destination, uint64_t signalByteSize)
{
	hipError_t hipError_t;

	hipError_t = hipMemcpy(*h_destination, d_signal, signalByteSize, hipMemcpyDeviceToHost);
	//copy data from host to device
	if( hipError_t != hipSuccess)
	{
		fprintf(stderr, "CopySignalToHost: error copying memory to the host\n");
		fprintf(stderr, "Cuda error code: %s\n", hipGetErrorString(hipError_t));
		//exit(1);
	}

}

