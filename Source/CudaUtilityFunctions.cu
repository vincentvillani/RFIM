/*
 * CudaUtilityFunctions.cu
 *
 *  Created on: 17/03/2016
 *      Author: vincentvillani
 */


#include "../Header/CudaUtilityFunctions.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

//Copies data from the host to the device and returns a device pointer
void CudaUtility_CopySignalToDevice(float* h_signal, float** d_destination, uint64_t signalByteSize)
{
	hipError_t hipError_t;

	hipError_t = hipMemcpy(*d_destination, h_signal, signalByteSize, hipMemcpyHostToDevice);

	//copy data from host to device
	if( hipError_t != hipSuccess)
	{
		fprintf(stderr, "CopySignalToDevice: error copying memory to the device\n");
		fprintf(stderr, "Cuda error code: %s\n", hipGetErrorString(hipError_t));
		//exit(1);
	}

}

//Copies data from the device to the host and returns a host pointer
void CudaUtility_CopySignalToHost(float* d_signal, float** h_destination, uint64_t signalByteSize)
{
	hipError_t hipError_t;

	hipError_t = hipMemcpy(*h_destination, d_signal, signalByteSize, hipMemcpyDeviceToHost);
	//copy data from host to device
	if( hipError_t != hipSuccess)
	{
		fprintf(stderr, "CopySignalToHost: error copying memory to the host\n");
		fprintf(stderr, "Cuda error code: %s\n", hipGetErrorString(hipError_t));
		//exit(1);
	}

}


float** CudaUtility_BatchAllocateDeviceArrays(uint32_t numberOfArrays, uint64_t arrayByteSize)
{
	//Allocate space for the pointers
	float** h_resultDevicePointers = (float**)malloc(sizeof(float*) * numberOfArrays);
	float** d_result;
	hipMalloc(&d_result, sizeof(float*) * numberOfArrays);


	//Allocate space for each pointer and copy it's address into h_resultDevicePointers + i
	for(uint32_t i = 0; i < numberOfArrays; ++i)
	{
		//Allocate each array's memory and store pointers to it on the host
		hipMalloc(&(h_resultDevicePointers[i]), arrayByteSize);
	}

	//Copy all the pointers into device memory
	hipMemcpy(d_result, h_resultDevicePointers, sizeof(float*) * numberOfArrays, hipMemcpyHostToDevice);

	free(h_resultDevicePointers);

	return d_result;
}



void CudaUtility_BatchDeallocateDeviceArrays(float** d_arrays, uint32_t numberOfArrays)
{
	//Copy the pointers to the host
	float** h_arraysDevicePointers = (float**)malloc(sizeof(float*) * numberOfArrays);
	hipMemcpy(h_arraysDevicePointers, d_arrays, sizeof(float*) * numberOfArrays, hipMemcpyDeviceToHost);

	for(uint32_t i = 0; i < numberOfArrays; ++i)
	{
		//Free each array
		hipFree(h_arraysDevicePointers[i]);
	}

	//Free the device pointers
	hipFree(d_arrays);

	//Free memory on the host
	free(h_arraysDevicePointers);
}


void CudaUtility_BatchCopyArraysHostToDevice(float** d_arrays, float** h_arrays, uint32_t numberOfArrays, uint64_t arrayByteSize)
{
	uint64_t pointersArrayByteSize = sizeof(float*) * numberOfArrays;

	//Copy the device pointers to the host
	float** h_devicePointers = (float**)malloc(pointersArrayByteSize);
	hipMemcpy(h_devicePointers, d_arrays, pointersArrayByteSize, hipMemcpyDeviceToHost);

	for(uint32_t i = 0; i < numberOfArrays; ++i)
	{
		//Copy the actual data across to each pointer
		hipMemcpy(h_devicePointers[i], h_arrays[i], arrayByteSize, hipMemcpyHostToDevice);
	}

	free(h_devicePointers);
}

