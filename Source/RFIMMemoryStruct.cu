
#include "../Header/RFIMMemoryStruct.h"

#include "../Header/CudaUtilityFunctions.h"

#include <stdio.h>


RFIMMemoryStruct* RFIMMemoryStructCreate(uint32_t h_valuesPerSample, uint32_t h_numberOfSamples, uint32_t h_dimensionToReduce, uint32_t h_batchSize)
{
	RFIMMemoryStruct* result = (RFIMMemoryStruct*)malloc(sizeof(RFIMMemoryStruct));

	result->cublasHandle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
	result->cusolverHandle = (hipsolverHandle_t*)malloc(sizeof(hipsolverHandle_t));

	hipblasStatus_t hipblasStatus_t;
	hipsolverStatus_t cusolverStatus;

	//Create the contexts for each library
	hipblasStatus_t = hipblasCreate( result->cublasHandle );
	cusolverStatus = hipsolverDnCreate( result->cusolverHandle );

	//Check the contexts started up ok
	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cublas context\n");
		//exit(1);
	}

	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cusolver context\n");
		//exit(1);
	}


	//Set signal attributes
	result->h_valuesPerSample = h_valuesPerSample;
	result->h_numberOfSamples = h_numberOfSamples;
	result->h_eigenVectorDimensionsToReduce = h_dimensionToReduce;
	result->h_batchSize = h_batchSize;



	//Setup the one vec
	//------------------------
	uint32_t oneVecByteSize = sizeof(float) * h_numberOfSamples;
	float* h_oneVec = (float*)malloc(oneVecByteSize);
	float** h_oneVecPointerArray = (float**)malloc(sizeof(float*) * h_batchSize);

	printf("0\n");

	//Fill the one vec with ones
	for(uint32_t i = 0; i < h_numberOfSamples; ++i)
	{
		h_oneVec[i] = 1;
	}

	printf("0.5\n");

	//Set each pointer to point to the same array
	for(uint32_t i = 0; i < h_batchSize; ++i)
	{
		h_oneVecPointerArray[i] = h_oneVec;
	}

	printf("0.75\n");


	//Allocate one array on the device, everything in the pointer array will point to this
	//float* d_oneVec;
	//hipMalloc(&d_oneVec, oneVecByteSize);
	//hipMemcpy(d_oneVec, h_oneVec, oneVecByteSize, hipMemcpyHostToDevice);



	uint64_t meanVecByteSize = sizeof(float) * h_valuesPerSample;
	uint64_t covarianceMatrixByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint64_t UByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint64_t SByteSize = sizeof(float) * h_valuesPerSample;
	uint64_t VTByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint32_t devInfoByteSize = sizeof(int);
	//Ask cusolver for the needed buffer size
	result->h_eigWorkingSpaceLength = 0;
	cusolverStatus = hipsolverDnSgesvd_bufferSize(*result->cusolverHandle, h_valuesPerSample, h_valuesPerSample, &(result->h_eigWorkingSpaceLength));
	//Check if it went well
	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error finding eigenvalue working buffer size\n");
		//exit(1);
	}
	uint32_t projectedSignalMatrixByteSize = sizeof(float) * ((h_valuesPerSample - result->h_eigenVectorDimensionsToReduce) * h_numberOfSamples);

	printf("1\n");

	//Allocate 2D pointers on the device
	result->d_oneVec = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, oneVecByteSize);
	CudaUtility_BatchCopyArraysHostToDevice(result->d_oneVec, h_oneVecPointerArray, h_batchSize, oneVecByteSize); //Copy the oneVec data to the 2D array

	result->d_meanVec = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, meanVecByteSize);

	printf("2\n");


	//Free memory
	//-----------------------------
	free(h_oneVec);
	free(h_oneVecPointerArray);

	printf("3\n");

	return result;
}



void RFIMMemoryStructDestroy(RFIMMemoryStruct* RFIMStruct)
{
	//Destroy the cuda library contexts
	hipblasDestroy(*RFIMStruct->cublasHandle);
	hipsolverDnDestroy(*RFIMStruct->cusolverHandle);

	free(RFIMStruct->cublasHandle);
	free(RFIMStruct->cusolverHandle);

	//Free the device array in on the GPU for the one vec, once (all pointers point to the same array)
	hipFree(RFIMStruct->d_oneVec[0]);


	//Free all batched arrays
	for(uint32_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		hipFree(RFIMStruct->d_meanVec[i]);
		hipFree(RFIMStruct->d_covarianceMatrix[i]);
		hipFree(RFIMStruct->d_U[i]);
		hipFree(RFIMStruct->d_S[i]);
		hipFree(RFIMStruct->d_VT[i]);
		hipFree(RFIMStruct->d_devInfo[i]);
		hipFree(RFIMStruct->d_projectedSignalMatrix[i]);
	}


	//Free arrays of pointers
	hipFree(RFIMStruct->d_oneVec); //Free the array of pointers
	hipFree(RFIMStruct->d_meanVec);
	hipFree(RFIMStruct->d_covarianceMatrix);
	hipFree(RFIMStruct->d_U);
	hipFree(RFIMStruct->d_S);
	hipFree(RFIMStruct->d_VT);
	hipFree(RFIMStruct->d_devInfo);
	hipFree(RFIMStruct->d_projectedSignalMatrix);


	//Deallocate the struct memory on the host
	free(RFIMStruct);

}


