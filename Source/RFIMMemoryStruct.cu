
#include "../Header/RFIMMemoryStruct.h"

#include "../Header/CudaUtilityFunctions.h"

#include <stdio.h>


RFIMMemoryStruct* RFIMMemoryStructCreate(uint32_t h_valuesPerSample, uint32_t h_numberOfSamples, uint32_t h_dimensionToReduce, uint32_t h_batchSize)
{
	RFIMMemoryStruct* result = (RFIMMemoryStruct*)malloc(sizeof(RFIMMemoryStruct));

	result->cublasHandle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
	result->cusolverHandle = (hipsolverHandle_t*)malloc(sizeof(hipsolverHandle_t));

	hipblasStatus_t hipblasStatus_t;
	hipsolverStatus_t cusolverStatus;

	//Create the contexts for each library
	hipblasStatus_t = hipblasCreate( result->cublasHandle );
	cusolverStatus = hipsolverDnCreate( result->cusolverHandle );

	//Check the contexts started up ok
	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cublas context\n");
		//exit(1);
	}

	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cusolver context\n");
		//exit(1);
	}


	//Set signal attributes
	result->h_valuesPerSample = h_valuesPerSample;
	result->h_numberOfSamples = h_numberOfSamples;
	result->h_eigenVectorDimensionsToReduce = h_dimensionToReduce;
	result->h_batchSize = h_batchSize;



	//Setup the one vec
	//------------------------
	uint32_t oneVecByteSize = sizeof(float) * h_numberOfSamples;
	float* h_oneVec = (float*)malloc(oneVecByteSize);

	//Fill the one vec with ones
	for(uint32_t i = 0; i < h_numberOfSamples; ++i)
	{
		h_oneVec[i] = 1;
	}

	//Allocate the pointers to the array
	hipMalloc(&result->d_oneVec, sizeof(float*) * h_batchSize);

	//Allocate one array on the device, everything in the pointer array will point to this
	float* d_oneVec;
	hipMalloc(&d_oneVec, oneVecByteSize);
	hipMemcpy(d_oneVec, h_oneVec, oneVecByteSize, hipMemcpyHostToDevice);

	//Allocate space for the pointers
	//------------------------
	hipMalloc(&(result->d_meanVec),  sizeof(float*) * h_batchSize);
	hipMalloc(&(result->d_covarianceMatrix), sizeof(float*) * h_batchSize);
	hipMalloc(&(result->d_U), sizeof(float*) * h_batchSize);
	hipMalloc(&(result->d_S), sizeof(float*) * h_batchSize);
	hipMalloc(&(result->d_VT), sizeof(float*) * h_batchSize);
	hipMalloc(&(result->d_devInfo), sizeof(int*) * h_batchSize);
	hipMalloc(&(result->d_eigWorkingSpace), sizeof(float*) * h_batchSize);
	hipMalloc(&(result->d_projectedSignalMatrix), sizeof(float*) * h_batchSize);


	uint32_t meanVecByteSize = sizeof(float) * h_valuesPerSample;
	uint32_t covarianceMatrixByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint32_t UByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint32_t SByteSize = sizeof(float) * h_valuesPerSample;
	uint32_t VTByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint32_t devInfoByteSize = sizeof(int);
	//Ask cusolver for the needed buffer size
	result->h_eigWorkingSpaceLength = 0;
	cusolverStatus = hipsolverDnSgesvd_bufferSize(*result->cusolverHandle, h_valuesPerSample, h_valuesPerSample, &(result->h_eigWorkingSpaceLength));
	//Check if it went well
	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error finding eigenvalue working buffer size\n");
		//exit(1);
	}
	uint32_t projectedSignalMatrixByteSize = sizeof(float) * ((h_valuesPerSample - result->h_eigenVectorDimensionsToReduce) * h_numberOfSamples);


	//set/allocate memory for all batching arrays
	//------------------------
	for(uint32_t i = 0; i < h_batchSize; ++i)
	{
		result->d_oneVec[i] = d_oneVec;

		hipMalloc(&(result->d_meanVec[i]), meanVecByteSize);
		hipMalloc(&(result->d_covarianceMatrix[i]), covarianceMatrixByteSize);
		hipMalloc(&(result->d_U[i]), UByteSize);
		hipMalloc(&(result->d_S[i]), SByteSize);
		hipMalloc(&(result->d_VT[i]), VTByteSize);
		hipMalloc(&(result->d_devInfo[i]), devInfoByteSize);
		hipMalloc(&(result->d_eigWorkingSpace[i]), result->h_eigWorkingSpaceLength);
		hipMalloc(&(result->d_projectedSignalMatrix[i]), projectedSignalMatrixByteSize);
	}

	//Free memory
	//-----------------------------
	free(h_oneVec);






	/*




	hipMalloc(&(result->d_projectedSignalMatrix),
			sizeof(float) * (h_valuesPerSample - result->h_eigenVectorDimensionsToReduce) * h_numberOfSamples);
	hipMemset(result->d_projectedSignalMatrix, 0,
			sizeof(float) * (h_valuesPerSample - result->h_eigenVectorDimensionsToReduce) * h_numberOfSamples);

				//Allocate memory for the reduced Eigenvector matrix and it's transpose
	hipMalloc(&(result->d_reducedEigenVecMatrix), sizeof(float) * h_valuesPerSample *
			(h_valuesPerSample - result->h_eigenVectorDimensionsToReduce));
	hipMemset(result->d_reducedEigenVecMatrix, 0, sizeof(float) * h_valuesPerSample *
			(h_valuesPerSample - result->h_eigenVectorDimensionsToReduce));

	*/

	return result;
}



void RFIMMemoryStructDestroy(RFIMMemoryStruct* RFIMStruct)
{
	//Destroy the cuda library contexts
	hipblasDestroy(*RFIMStruct->cublasHandle);
	hipsolverDnDestroy(*RFIMStruct->cusolverHandle);

	free(RFIMStruct->cublasHandle);
	free(RFIMStruct->cusolverHandle);

	//Free the device array in on the GPU for the one vec, once (all pointers point to the same array)
	hipFree(RFIMStruct->d_oneVec[0]);


	//Free all batched arrays
	for(uint32_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		hipFree(RFIMStruct->d_meanVec[i]);
		hipFree(RFIMStruct->d_covarianceMatrix[i]);
		hipFree(RFIMStruct->d_U[i]);
		hipFree(RFIMStruct->d_S[i]);
		hipFree(RFIMStruct->d_VT[i]);
		hipFree(RFIMStruct->d_devInfo[i]);
		hipFree(RFIMStruct->d_projectedSignalMatrix[i]);
	}


	//Free arrays of pointers
	hipFree(RFIMStruct->d_oneVec); //Free the array of pointers
	hipFree(RFIMStruct->d_meanVec);
	hipFree(RFIMStruct->d_covarianceMatrix);
	hipFree(RFIMStruct->d_U);
	hipFree(RFIMStruct->d_S);
	hipFree(RFIMStruct->d_VT);
	hipFree(RFIMStruct->d_devInfo);
	hipFree(RFIMStruct->d_projectedSignalMatrix);

	/*

	//Deallocate the mean working memory
	hipFree(RFIMStruct->d_oneVec);
	hipFree(RFIMStruct->d_meanVec);
	//hipFree(RFIMStruct->d_meanMatrix);

	//Deallocate covariance working memory
	hipFree(RFIMStruct->d_upperTriangularCovarianceMatrix);
	hipFree(RFIMStruct->d_upperTriangularTransposedMatrix);
	hipFree(RFIMStruct->d_fullSymmetricCovarianceMatrix);

	//Deallocate eigenvector/value working memory
	hipFree(RFIMStruct->d_U);
	hipFree(RFIMStruct->d_S);
	hipFree(RFIMStruct->d_VT);
	hipFree(RFIMStruct->d_devInfo);
	hipFree(RFIMStruct->d_eigWorkingSpace);

	hipFree(RFIMStruct->d_reducedEigenVecMatrix);

	hipFree(RFIMStruct->d_projectedSignalMatrix);

	*/

	//Deallocate the struct memory on the host
	free(RFIMStruct);

}


