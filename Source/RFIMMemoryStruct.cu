
#include "../Header/RFIMMemoryStruct.h"

#include "../Header/CudaUtilityFunctions.h"

#include <stdio.h>


RFIMMemoryStruct* RFIMMemoryStructCreate(uint64_t h_valuesPerSample, uint64_t h_numberOfSamples, uint64_t h_dimensionToReduce,
		uint64_t h_batchSize, uint32_t threadIndex)
{

	RFIMMemoryStruct* result;
	hipHostMalloc(&result, sizeof(RFIMMemoryStruct));

	hipHostMalloc(&(result->cublasHandle), sizeof(hipblasHandle_t));

	hipHostMalloc(&(result->cusolverHandle), sizeof(hipsolverHandle_t));

	hipblasStatus_t hipblasStatus_t;
	hipsolverStatus_t cusolverStatus;

	//Start up the cudaStream
	hipStreamCreateWithFlags(&result->cudaStream, hipStreamNonBlocking);

	//Create the contexts for each library
	hipblasStatus_t = hipblasCreate( result->cublasHandle );
	cusolverStatus = hipsolverDnCreate( result->cusolverHandle );

	//Have the library handles execute on this newly created stream
	hipblasSetStream(*result->cublasHandle, result->cudaStream);
	hipsolverSetStream(*result->cusolverHandle, result->cudaStream);

	//Check the contexts started up ok
	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cublas context\n");
		exit(1);
	}

	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cusolver context\n");
		exit(1);
	}


	//Set signal attributes
	result->h_valuesPerSample = h_valuesPerSample;
	result->h_numberOfSamples = h_numberOfSamples;
	result->h_eigenVectorDimensionsToReduce = h_dimensionToReduce;
	result->h_batchSize = h_batchSize;
	result->h_threadId = threadIndex;



	//Setup the one vec
	//------------------------
	uint64_t oneVecByteSize = sizeof(float) * h_numberOfSamples;

	float* h_oneVec;
	hipHostMalloc(&h_oneVec, oneVecByteSize);

	float** h_oneVecPointerArray;
	hipHostMalloc(&h_oneVecPointerArray, sizeof(float*) * h_batchSize);


	//Fill the one vec with ones
	for(uint64_t i = 0; i < h_numberOfSamples; ++i)
	{
		h_oneVec[i] = 1;
	}


	//Set each pointer to point to the same array
	for(uint64_t i = 0; i < h_batchSize; ++i)
	{
		h_oneVecPointerArray[i] = h_oneVec;
	}



	uint64_t meanVecByteSize = sizeof(float) * h_valuesPerSample;
	uint64_t covarianceMatrixByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint64_t UByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint64_t SByteSize = sizeof(float) * h_valuesPerSample;
	uint64_t VTByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;

	//Ask cusolver for the needed buffer size
	result->h_eigWorkingSpaceLength = 0;
	cusolverStatus = hipsolverDnSgesvd_bufferSize(*result->cusolverHandle, h_valuesPerSample, h_valuesPerSample, &(result->h_eigWorkingSpaceLength));
	//Check if it went well
	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error finding eigenvalue working buffer size\n");
		exit(1);
	}
	uint64_t projectedSignalMatrixByteSize = sizeof(float) * h_valuesPerSample * h_numberOfSamples;



	//Allocate 2D pointers on the device
	result->d_oneVec = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, oneVecByteSize, &(result->cudaStream));
	CudaUtility_BatchCopyArraysHostToDevice(result->d_oneVec, h_oneVecPointerArray, h_batchSize, oneVecByteSize, &(result->cudaStream)); //Copy the oneVec data to the 2D array

	result->d_meanVec = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, meanVecByteSize, &(result->cudaStream));

	result->d_covarianceMatrix = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, covarianceMatrixByteSize, &(result->cudaStream));
	result->d_U = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, UByteSize, &(result->cudaStream));
	result->d_S = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, SByteSize, &(result->cudaStream));
	result->d_VT = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, VTByteSize, &(result->cudaStream));
	result->d_eigWorkingSpace = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, result->h_eigWorkingSpaceLength, &(result->cudaStream));
	hipMalloc(&(result->d_devInfo), sizeof(int) * h_batchSize);
	//result->h_devInfoValues = (int*)malloc(sizeof(int) * h_batchSize);
	hipHostMalloc(&(result->h_devInfoValues), sizeof(int) * h_batchSize);
	result->d_projectedSignalMatrix = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, projectedSignalMatrixByteSize, &(result->cudaStream));



	//Allocate space for the pointers to device memory, this is used to speed up the eigenvector solver part of the RFIM
	uint64_t pointersArrayByteSize = sizeof(float*) * h_batchSize;

	hipHostMalloc(&(result->h_covarianceMatrixDevicePointers), pointersArrayByteSize);
	hipHostMalloc(&(result->h_UDevicePointers), pointersArrayByteSize); //Allocate pinned memory for use with async memcpy
	hipHostMalloc(&(result->h_SDevicePointers), pointersArrayByteSize);
	hipHostMalloc(&(result->h_VTDevicePointers), pointersArrayByteSize);
	hipHostMalloc(&(result->h_eigWorkingSpaceDevicePointers), pointersArrayByteSize);


	//Copy the pointers to device memory over to the host memory
	hipMemcpyAsync(result->h_covarianceMatrixDevicePointers, result->d_covarianceMatrix, pointersArrayByteSize, hipMemcpyDeviceToHost, result->cudaStream);
	hipMemcpyAsync(result->h_UDevicePointers, result->d_U, pointersArrayByteSize, hipMemcpyDeviceToHost, result->cudaStream);
	hipMemcpyAsync(result->h_SDevicePointers, result->d_S, pointersArrayByteSize, hipMemcpyDeviceToHost, result->cudaStream);
	hipMemcpyAsync(result->h_VTDevicePointers, result->d_VT, pointersArrayByteSize, hipMemcpyDeviceToHost, result->cudaStream);
	hipMemcpyAsync(result->h_eigWorkingSpaceDevicePointers, result->d_eigWorkingSpace, pointersArrayByteSize, hipMemcpyDeviceToHost, result->cudaStream);


	//Wait for all memcopies, memsets etc to occur
	hipStreamSynchronize(result->cudaStream);

	//Free memory
	//-----------------------------
	hipHostFree(h_oneVec);
	hipHostFree(h_oneVecPointerArray);


	//Check for errors
	hipError_t hipError_t = hipGetLastError();

	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "RFIMMemoryStructCreate: Probably failed to allocate memory\n");
		exit(1);
	}


	return result;
}



void RFIMMemoryStructDestroy(RFIMMemoryStruct* RFIMStruct)
{


	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_oneVec, RFIMStruct->h_batchSize, &(RFIMStruct->cudaStream));
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_meanVec, RFIMStruct->h_batchSize, &(RFIMStruct->cudaStream));
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_covarianceMatrix, RFIMStruct->h_batchSize, &(RFIMStruct->cudaStream));
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_U, RFIMStruct->h_batchSize, &(RFIMStruct->cudaStream));
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_S, RFIMStruct->h_batchSize, &(RFIMStruct->cudaStream));
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_VT, RFIMStruct->h_batchSize, &(RFIMStruct->cudaStream));
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_eigWorkingSpace, RFIMStruct->h_batchSize, &(RFIMStruct->cudaStream));
	hipFree(RFIMStruct->d_devInfo);
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_projectedSignalMatrix, RFIMStruct->h_batchSize, &(RFIMStruct->cudaStream));

	//Free the host pointers to device memory
	hipHostFree(RFIMStruct->h_covarianceMatrixDevicePointers);
	hipHostFree(RFIMStruct->h_UDevicePointers);
	hipHostFree(RFIMStruct->h_SDevicePointers);
	hipHostFree(RFIMStruct->h_VTDevicePointers);
	hipHostFree(RFIMStruct->h_eigWorkingSpaceDevicePointers);
	hipHostFree(RFIMStruct->h_devInfoValues);

	//Destroy the cuda library contexts
	hipblasDestroy(*RFIMStruct->cublasHandle);
	hipsolverDnDestroy(*RFIMStruct->cusolverHandle);


	hipHostFree(RFIMStruct->cublasHandle);
	hipHostFree(RFIMStruct->cusolverHandle);

	hipStreamDestroy(RFIMStruct->cudaStream);

	//Deallocate the struct memory on the host
	hipHostFree(RFIMStruct);

}


