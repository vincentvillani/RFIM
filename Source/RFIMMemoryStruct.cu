
#include "../Header/RFIMMemoryStruct.h"

#include "../Header/CudaUtilityFunctions.h"

#include <stdio.h>


RFIMMemoryStruct* RFIMMemoryStructCreate(uint32_t h_valuesPerSample, uint32_t h_numberOfSamples)
{
	RFIMMemoryStruct* result = (RFIMMemoryStruct*)malloc(sizeof(RFIMMemoryStruct));

	result->cublasHandle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
	result->cusolverHandle = (hipsolverHandle_t*)malloc(sizeof(hipsolverHandle_t));

	hipblasStatus_t hipblasStatus_t;
	hipsolverStatus_t cusolverStatus;

	//Create the contexts for each library
	hipblasStatus_t = hipblasCreate( result->cublasHandle );
	cusolverStatus = hipsolverDnCreate( result->cusolverHandle );

	//Check the contexts started up ok
	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cublas context\n");
		//exit(1);
	}

	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cusolver context\n");
		//exit(1);
	}


	//Set signal attributes
	result->h_valuesPerSample = h_valuesPerSample;
	result->h_numberOfSamples = h_numberOfSamples;

	//Allocate memory for the filtered signal
	//Set the original signal to NULL originally
	//result->d_originalSignal = NULL; //The RFIM routine will exit if this is set to NULL upon entry
	//hipMalloc(&(result->d_filteredSignal), sizeof(float) * h_valuesPerSample * h_numberOfSamples);


	//Setup the mean working memory
	float* h_oneVec = (float*)malloc(sizeof(float) * h_valuesPerSample);

	//Fill the one vec with ones
	for(uint32_t i = 0; i < h_valuesPerSample; ++i)
	{
		h_oneVec[i] = 1;
	}


	hipMalloc(&result->d_oneVec, sizeof(float) * h_valuesPerSample);
	hipMemset(result->d_oneVec, 0, sizeof(float) * h_valuesPerSample);

	//Copy the one vec to the device
	CudaUtility_CopySignalToDevice(h_oneVec, &result->d_oneVec,  sizeof(float) * h_valuesPerSample);

	//Free the host memory, don't need it anymore
	free(h_oneVec);



	//Allocate working space for the other mean
	hipMalloc(&(result->d_meanVec), sizeof(float) * h_valuesPerSample);
	hipMemset(result->d_meanVec, 0, sizeof(float) * h_valuesPerSample);
	//hipMalloc(&(result->d_meanMatrix), sizeof(float) * h_valuesPerSample * h_valuesPerSample);


	//Allocate space for the covariance matrix
	hipMalloc(&(result->d_upperTriangularCovarianceMatrix), sizeof(float) * h_valuesPerSample * h_valuesPerSample);
	hipMemset(result->d_upperTriangularCovarianceMatrix, 0, sizeof(float) * h_valuesPerSample * h_valuesPerSample);

	hipMalloc(&(result->d_upperTriangularTransposedMatrix), sizeof(float) * h_valuesPerSample * h_valuesPerSample);
	hipMemset(result->d_upperTriangularTransposedMatrix, 0, sizeof(float) * h_valuesPerSample * h_valuesPerSample);

	hipMalloc(&(result->d_fullSymmetricCovarianceMatrix), sizeof(float) * h_valuesPerSample * h_valuesPerSample);
	hipMemset(result->d_fullSymmetricCovarianceMatrix, 0, sizeof(float) * h_valuesPerSample * h_valuesPerSample);



	//Allocate working space for the eigenvector/value solver
	hipMalloc(&(result->d_U), sizeof(float) * h_valuesPerSample * h_valuesPerSample);
	hipMemset(result->d_U, 0, sizeof(float) * h_valuesPerSample * h_valuesPerSample);


	hipMalloc(&(result->d_S), sizeof(float) * h_valuesPerSample);
	hipMemset(result->d_S, 0, sizeof(float) * h_valuesPerSample);


	hipMalloc(&(result->d_VT), sizeof(float) * h_valuesPerSample * h_valuesPerSample);
	hipMemset(result->d_VT, 0, sizeof(float) * h_valuesPerSample * h_valuesPerSample);

	hipMalloc(&(result->d_devInfo), sizeof(int));
	hipMemset(result->d_devInfo, 0, sizeof(int));


	//Ask cusolver for the needed buffer size
	result->h_eigWorkingSpaceLength = 0;

	cusolverStatus = hipsolverDnSgesvd_bufferSize(*result->cusolverHandle, h_valuesPerSample, h_valuesPerSample, &(result->h_eigWorkingSpaceLength));

	//Check if it went well
	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error finding eigenvalue working buffer size\n");
		//exit(1);
	}

	//Allocate memory for it
	hipMalloc( &(result->d_eigWorkingSpace), result->h_eigWorkingSpaceLength);
	hipMemset(result->d_eigWorkingSpace, 0, result->h_eigWorkingSpaceLength);


	//Eigenvectors dimensions to reduce, chosen arbitrarily for now
	//TODO: Come back to this. This will probably change
	result->h_eigenVectorDimensionsToReduce = 2;

	//Allocate memory for the reduced Eigenvector matrix and it's transpose
	hipMalloc(&(result->d_reducedEigenVecMatrix), sizeof(float) * h_valuesPerSample *
			(h_valuesPerSample - result->h_eigenVectorDimensionsToReduce));
	hipMemset(result->d_reducedEigenVecMatrix, 0, sizeof(float) * h_valuesPerSample *
			(h_valuesPerSample - result->h_eigenVectorDimensionsToReduce));



	hipMalloc(&(result->d_reducedEigenVecMatrixTranspose), sizeof(float) * h_valuesPerSample *
			(h_valuesPerSample - result->h_eigenVectorDimensionsToReduce));
	hipMemset(result->d_reducedEigenVecMatrixTranspose, 0, sizeof(float) * h_valuesPerSample *
			(h_valuesPerSample - result->h_eigenVectorDimensionsToReduce));

	//Outer product returns the matrix back to it's original dimensionality
	hipMalloc(&(result->d_reducedEigenMatrixOuterProduct), sizeof(float) * h_valuesPerSample * h_valuesPerSample);
	hipMemset(result->d_reducedEigenMatrixOuterProduct, 0, sizeof(float) * h_valuesPerSample *
				(h_valuesPerSample - result->h_eigenVectorDimensionsToReduce));


	return result;
}



void RFIMMemoryStructDestroy(RFIMMemoryStruct* RFIMStruct)
{
	//Destroy the cuda library contexts
	hipblasDestroy(*RFIMStruct->cublasHandle);
	hipsolverDnDestroy(*RFIMStruct->cusolverHandle);

	free(RFIMStruct->cublasHandle);
	free(RFIMStruct->cusolverHandle);

	//Deallocate the mean working memory
	hipFree(RFIMStruct->d_oneVec);
	hipFree(RFIMStruct->d_meanVec);
	//hipFree(RFIMStruct->d_meanMatrix);

	//Deallocate covariance working memory
	hipFree(RFIMStruct->d_upperTriangularCovarianceMatrix);
	hipFree(RFIMStruct->d_upperTriangularTransposedMatrix);
	hipFree(RFIMStruct->d_fullSymmetricCovarianceMatrix);

	//Deallocate eigenvector/value working memory
	hipFree(RFIMStruct->d_U);
	hipFree(RFIMStruct->d_S);
	hipFree(RFIMStruct->d_VT);
	hipFree(RFIMStruct->d_devInfo);
	hipFree(RFIMStruct->d_eigWorkingSpace);

	hipFree(RFIMStruct->d_reducedEigenVecMatrix);
	hipFree(RFIMStruct->d_reducedEigenVecMatrixTranspose);
	hipFree(RFIMStruct->d_reducedEigenMatrixOuterProduct);



	//Deallocate the struct memory on the host
	free(RFIMStruct);

}


