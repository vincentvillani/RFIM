
#include "../Header/RFIMMemoryStruct.h"

#include "../Header/CudaUtilityFunctions.h"

#include <stdio.h>


RFIMMemoryStruct* RFIMMemoryStructCreate(uint64_t h_valuesPerSample, uint64_t h_numberOfSamples, uint64_t h_dimensionToReduce, uint64_t h_batchSize)
{
	RFIMMemoryStruct* result = (RFIMMemoryStruct*)malloc(sizeof(RFIMMemoryStruct));

	result->cublasHandle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
	result->cusolverHandle = (hipsolverHandle_t*)malloc(sizeof(hipsolverHandle_t));

	hipblasStatus_t hipblasStatus_t;
	hipsolverStatus_t cusolverStatus;

	//Create the contexts for each library
	hipblasStatus_t = hipblasCreate( result->cublasHandle );
	cusolverStatus = hipsolverDnCreate( result->cusolverHandle );

	//Check the contexts started up ok
	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cublas context\n");
		exit(1);
	}

	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cusolver context\n");
		exit(1);
	}


	//Set signal attributes
	result->h_valuesPerSample = h_valuesPerSample;
	result->h_numberOfSamples = h_numberOfSamples;
	result->h_eigenVectorDimensionsToReduce = h_dimensionToReduce;
	result->h_batchSize = h_batchSize;



	//Setup the one vec
	//------------------------
	uint64_t oneVecByteSize = sizeof(float) * h_numberOfSamples;
	float* h_oneVec = (float*)malloc(oneVecByteSize);
	float** h_oneVecPointerArray = (float**)malloc(sizeof(float*) * h_batchSize);


	//Fill the one vec with ones
	for(uint64_t i = 0; i < h_numberOfSamples; ++i)
	{
		h_oneVec[i] = 1;
	}


	//Set each pointer to point to the same array
	for(uint64_t i = 0; i < h_batchSize; ++i)
	{
		h_oneVecPointerArray[i] = h_oneVec;
	}



	//Allocate one array on the device, everything in the pointer array will point to this
	//float* d_oneVec;
	//hipMalloc(&d_oneVec, oneVecByteSize);
	//hipMemcpy(d_oneVec, h_oneVec, oneVecByteSize, hipMemcpyHostToDevice);



	uint64_t meanVecByteSize = sizeof(float) * h_valuesPerSample;
	uint64_t covarianceMatrixByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint64_t UByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;
	uint64_t SByteSize = sizeof(float) * h_valuesPerSample;
	uint64_t VTByteSize = sizeof(float) * h_valuesPerSample * h_valuesPerSample;

	//Ask cusolver for the needed buffer size
	result->h_eigWorkingSpaceLength = 0;
	cusolverStatus = hipsolverDnSgesvd_bufferSize(*result->cusolverHandle, h_valuesPerSample, h_valuesPerSample, &(result->h_eigWorkingSpaceLength));
	//Check if it went well
	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error finding eigenvalue working buffer size\n");
		exit(1);
	}
	uint64_t projectedSignalMatrixByteSize = sizeof(float) * h_valuesPerSample * h_numberOfSamples;



	//Allocate 2D pointers on the device
	result->d_oneVec = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, oneVecByteSize);
	CudaUtility_BatchCopyArraysHostToDevice(result->d_oneVec, h_oneVecPointerArray, h_batchSize, oneVecByteSize); //Copy the oneVec data to the 2D array

	result->d_meanVec = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, meanVecByteSize);

	result->d_covarianceMatrix = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, covarianceMatrixByteSize);
	result->d_U = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, UByteSize);
	result->d_S = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, SByteSize);
	result->d_VT = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, VTByteSize);
	result->d_eigWorkingSpace = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, result->h_eigWorkingSpaceLength);
	hipMalloc(&(result->d_devInfo), sizeof(int) * h_batchSize);
	result->h_devInfoValues = (int*)malloc(sizeof(int) * h_batchSize);
	result->d_projectedSignalMatrix = CudaUtility_BatchAllocateDeviceArrays(h_batchSize, projectedSignalMatrixByteSize);



	//Allocate space for the pointers to device memory, this is used to speed up the eigenvector solver part of the RFIM
	uint64_t pointersArrayByteSize = sizeof(float*) * h_batchSize;

	result->h_covarianceMatrixDevicePointers = (float**)malloc(pointersArrayByteSize);
	result->h_UDevicePointers = (float**)malloc(pointersArrayByteSize);
	//result->h_UDeviceOffsetPointers = (float**)malloc(pointersArrayByteSize);
	result->h_SDevicePointers = (float**)malloc(pointersArrayByteSize);
	result->h_VTDevicePointers = (float**)malloc(pointersArrayByteSize);
	result->h_eigWorkingSpaceDevicePointers = (float**)malloc(pointersArrayByteSize);


	//Copy the pointers to device memory over to the host memory
	hipMemcpy(result->h_covarianceMatrixDevicePointers, result->d_covarianceMatrix, pointersArrayByteSize, hipMemcpyDeviceToHost);
	hipMemcpy(result->h_UDevicePointers, result->d_U, pointersArrayByteSize, hipMemcpyDeviceToHost);
	hipMemcpy(result->h_SDevicePointers, result->d_S, pointersArrayByteSize, hipMemcpyDeviceToHost);
	hipMemcpy(result->h_VTDevicePointers, result->d_VT, pointersArrayByteSize, hipMemcpyDeviceToHost);
	hipMemcpy(result->h_eigWorkingSpaceDevicePointers, result->d_eigWorkingSpace, pointersArrayByteSize, hipMemcpyDeviceToHost);




	//Free memory
	//-----------------------------
	free(h_oneVec);
	free(h_oneVecPointerArray);

	hipError_t hipError_t = hipGetLastError();

	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "RFIMMemoryStructCreate: Probably failed to allocate memory\n");
		exit(1);
	}


	return result;
}



void RFIMMemoryStructDestroy(RFIMMemoryStruct* RFIMStruct)
{
	//Destroy the cuda library contexts
	hipblasDestroy(*RFIMStruct->cublasHandle);
	hipsolverDnDestroy(*RFIMStruct->cusolverHandle);

	free(RFIMStruct->cublasHandle);
	free(RFIMStruct->cusolverHandle);

	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_oneVec, RFIMStruct->h_batchSize);
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_meanVec, RFIMStruct->h_batchSize);
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_covarianceMatrix, RFIMStruct->h_batchSize);
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_U, RFIMStruct->h_batchSize);
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_S, RFIMStruct->h_batchSize);
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_VT, RFIMStruct->h_batchSize);
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_eigWorkingSpace, RFIMStruct->h_batchSize);
	hipFree(RFIMStruct->d_devInfo);
	CudaUtility_BatchDeallocateDeviceArrays(RFIMStruct->d_projectedSignalMatrix, RFIMStruct->h_batchSize);

	//Free the host pointers to device memory
	free(RFIMStruct->h_covarianceMatrixDevicePointers);
	free(RFIMStruct->h_UDevicePointers);
	//free(RFIMStruct->h_UDeviceOffsetPointers);
	free(RFIMStruct->h_SDevicePointers);
	free(RFIMStruct->h_VTDevicePointers);
	free(RFIMStruct->h_eigWorkingSpaceDevicePointers);
	free(RFIMStruct->h_devInfoValues);

	//Deallocate the struct memory on the host
	free(RFIMStruct);

}


