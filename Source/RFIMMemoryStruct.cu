
#include "../Header/RFIMMemoryStruct.h"

#include "../Header/CudaUtilityFunctions.h"

#include <stdio.h>


RFIMMemoryStruct* RFIMMemoryStructCreate(uint64_t h_valuesPerSample, uint64_t h_numberOfSamples, uint64_t h_dimensionToReduce,
		uint64_t h_batchSize, uint64_t h_numberOfCUDAStreams)
{

	RFIMMemoryStruct* result;
	hipHostMalloc(&result, sizeof(RFIMMemoryStruct));




	//Set signal attributes
	//------------------------
	result->h_valuesPerSample = h_valuesPerSample;
	result->h_numberOfSamples = h_numberOfSamples;
	result->h_eigenVectorDimensionsToReduce = h_dimensionToReduce;
	result->h_batchSize = h_batchSize;
	result->h_cudaStreamsLength = h_numberOfCUDAStreams;



	//Setup library handles
	//------------------------
	hipHostMalloc(&(result->cublasHandle), sizeof(hipblasHandle_t));
	hipHostMalloc(&(result->cusolverHandle), sizeof(hipsolverHandle_t));

	hipblasStatus_t hipblasStatus_t;
	hipsolverStatus_t cusolverStatus;

	//Create the contexts for each library
	hipblasStatus_t = hipblasCreate( result->cublasHandle );
	cusolverStatus = hipsolverDnCreate( result->cusolverHandle );

	//Check the contexts started up ok
	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cublas context\n");
		exit(1);
	}

	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{
		fprintf(stderr, "RFIMMemory::RFIMMemory(): Error starting cusolver context\n");
		exit(1);
	}


	//Allocate space for the cudaSteams
	hipHostMalloc(&(result->h_cudaStreams), sizeof(hipStream_t) * h_numberOfCUDAStreams);
	for(uint64_t i = 0; i < h_numberOfCUDAStreams; ++i)
	{
		hipStreamCreateWithFlags(result->h_cudaStreams + i, hipStreamNonBlocking);
	}





	//Setup the one vec, we use the same memory over and over again, it should never change
	//------------------------
	uint64_t oneVecLength = h_numberOfSamples;
	uint64_t oneVecByteSize = sizeof(float) * oneVecLength;


	float* h_oneVec;
	hipHostMalloc(&h_oneVec, oneVecByteSize);
	hipMalloc(&(result->d_oneVec), oneVecByteSize);

	//Fill the one vec with ones
	for(uint64_t i = 0; i < oneVecLength; ++i)
	{
		h_oneVec[i] = 1;
	}

	//copy the ones over and free the host memory
	hipMemcpy(result->d_oneVec, h_oneVec, oneVecByteSize, hipMemcpyHostToDevice);
	hipHostFree(h_oneVec);





	//Setup the mean vec
	//------------------------
	uint64_t meanVecLength = h_valuesPerSample * h_batchSize;
	uint64_t meanVecByteSize = sizeof(float) * meanVecLength;

	result->h_meanVecBatchOffset = h_valuesPerSample;

	hipMalloc(&(result->d_meanVec), meanVecByteSize);





	//Setup the covariance matrix
	uint64_t covarianceMatrixLength = h_valuesPerSample * h_valuesPerSample * h_batchSize;
	uint64_t covarianceMatrixByteSize = sizeof(float) * covarianceMatrixLength;

	result->h_covarianceMatrixBatchOffset = h_valuesPerSample * h_valuesPerSample;

	hipMalloc(&(result->d_covarianceMatrix), covarianceMatrixByteSize);


	return result;
}



void RFIMMemoryStructDestroy(RFIMMemoryStruct* RFIMStruct)
{
	//Free device memory
	hipFree(RFIMStruct->d_oneVec);
	hipFree(RFIMStruct->d_meanVec);
	hipFree(RFIMStruct->d_covarianceMatrix);


	//Destroy the cuda library contexts
	hipblasDestroy(*RFIMStruct->cublasHandle);
	hipsolverDnDestroy(*RFIMStruct->cusolverHandle);


	hipHostFree(RFIMStruct->cublasHandle);
	hipHostFree(RFIMStruct->cusolverHandle);

	//Destroy the cuda streams
	for(uint64_t i = 0; i < RFIMStruct->h_cudaStreamsLength; ++i)
	{
		hipStreamDestroy(RFIMStruct->h_cudaStreams[i]);
	}
	hipHostFree(RFIMStruct->h_cudaStreams);

	//Deallocate the struct memory on the host
	hipHostFree(RFIMStruct);

}


