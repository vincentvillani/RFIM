/*
 * RFIMHelperFunctions.cu
 *
 *  Created on: 17/03/2016
 *      Author: vincentvillani
 */

#include "../Header/RFIMHelperFunctions.h"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

float* Device_GenerateWhiteNoiseSignal(uint64_t h_valuesPerSample, uint64_t h_numberOfSamples)
{
	uint64_t totalSignalLength = h_valuesPerSample * h_numberOfSamples;
	uint64_t totalSignalByteSize = totalSignalLength * sizeof(float);

	float* d_signal;

	hipError_t error;

	//Allocate the memory required to store the signal
	error =  hipMalloc(&d_signal, totalSignalByteSize);

	//Check that it was allocated successfully
	if(error != hipSuccess)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Unable to allocate %llu bytes of memory on the device\n", totalSignalByteSize);
		exit(1);
	}

	//Start cuda rand library
	hiprandGenerator_t rngGen;

	if( hiprandCreateGenerator(&rngGen, HIPRAND_RNG_PSEUDO_DEFAULT) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Unable to start cuRand library\n");
		exit(1);
	}


	//Set the RNG seed
	if((hiprandSetPseudoRandomGeneratorSeed(rngGen, 1)) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Unable to set the RNG Seed value\n");
		exit(1);
	}


	//Generate the signal!
	//Generate random numbers on the device
	//Generate random numbers using a normal distribution
	//Normal distribution should emulate white noise hopefully?
	//Generate signal
	if(hiprandGenerateNormal(rngGen, d_signal, totalSignalLength, 0.0f, 1.0f) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Error when generating the signal\n");
		exit(1);
	}



	//Destroy the RNG
	if(hiprandDestroyGenerator(rngGen) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Error in destroying the RNG generator \n");
		exit(1);
	}


	//Return the generated signal that resides in DEVICE memory
	return d_signal;

}

