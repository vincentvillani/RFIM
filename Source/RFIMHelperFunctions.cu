/*
 * RFIMHelperFunctions.cu
 *
 *  Created on: 17/03/2016
 *      Author: vincentvillani
 */

#include "../Header/RFIMHelperFunctions.h"

#include <stdio.h>
#include <string.h>
#include <mkl.h>
#include <mkl_lapacke.h>



#include "../Header/Kernels.h"
#include "../Header/CudaMacros.h"


//Private helper functions
//--------------------------





//Private functions implementation
//----------------------------------




//--------------------------


float* Device_GenerateWhiteNoiseSignal(hiprandGenerator_t* rngGen, uint64_t h_valuesPerSample, uint64_t h_numberOfSamples, uint64_t h_batchSize)
{

	uint64_t totalSignalLength = h_valuesPerSample * h_numberOfSamples * h_batchSize;
	uint64_t totalSignalByteSize = sizeof(float) * totalSignalLength;


	float* d_signalMatrix;

	hipMalloc(&d_signalMatrix, totalSignalByteSize);

	//Generate the signal!
	//Generate random numbers on the device
	//Generate random numbers using a normal distribution
	//Normal distribution should emulate white noise hopefully?
	//Generate signal
	if(hiprandGenerateNormal(*rngGen, d_signalMatrix, totalSignalLength, 0.0f, 1.0f) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Error when generating the signal\n");
		exit(1);
	}


	hipDeviceSynchronize();

	return d_signalMatrix;

}


float* Device_GenerateWhiteNoiseSignal(hiprandGenerator_t* rngGen, uint64_t h_valuesPerSample, uint64_t h_numberOfSamples,
		uint64_t h_batchSize, uint64_t h_threadNum)
{

	/*
	HIPRAND_STATUS_NOT_INITIALIZED if the generator was never created \n
	 * - HIPRAND_STATUS_PREEXISTING_FAILURE if there was an existing error from
	 *    a previous kernel launch \n
	 * - HIPRAND_STATUS_LAUNCH_FAILURE if the kernel launch failed for any reason \n
	 * - HIPRAND_STATUS_LENGTH_NOT_MULTIPLE if the number of output samples is
	 *    not a multiple of the quasirandom dimension, or is not a multiple
	 *    of two for pseudorandom generators \n
	 * - HIPRAND_STATUS_SUCCESS if the results were generated successfully \n
	 * */


	uint64_t totalSignalLength = h_valuesPerSample * h_numberOfSamples * h_batchSize * h_threadNum;
	uint64_t totalSignalByteSize = sizeof(float) * totalSignalLength;


	float* d_signalMatrix;

	hipMalloc(&d_signalMatrix, totalSignalByteSize);

	//Generate the signal!
	//Generate random numbers on the device
	//Generate random numbers using a normal distribution
	//Normal distribution should emulate white noise hopefully?
	//Generate signal
	hiprandStatus_t hiprandStatus = hiprandGenerateNormal(*rngGen, d_signalMatrix, totalSignalLength, 0.0f, 1.0f);

	if(hiprandStatus != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Error when generating the signal\n");

		if(hiprandStatus == HIPRAND_STATUS_NOT_INITIALIZED)
			fprintf(stderr, "HIPRAND_STATUS_NOT_INITIALIZED\n");
		else if(hiprandStatus == HIPRAND_STATUS_PREEXISTING_FAILURE)
			fprintf(stderr, "HIPRAND_STATUS_PREEXISTING_FAILURE\n");
		else if(hiprandStatus == HIPRAND_STATUS_LAUNCH_FAILURE)
			fprintf(stderr, "HIPRAND_STATUS_LAUNCH_FAILURE\n");
		else if(hiprandStatus == HIPRAND_STATUS_LENGTH_NOT_MULTIPLE)
			fprintf(stderr, "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE\n");
		else
			fprintf(stderr, "Unknown cuRand error\n");


		exit(1);
	}


	hipDeviceSynchronize();

	return d_signalMatrix;

}


/*
float* Device_GenerateWhiteNoiseSignal(hiprandGenerator_t* rngGen, uint64_t h_valuesPerSample, uint64_t h_numberOfSamples,
		uint64_t h_batchSize, uint64_t h_threadNum, float mean, float stdDev)
{
	uint64_t totalSignalLength = h_valuesPerSample * h_numberOfSamples * h_batchSize * h_threadNum;
	uint64_t totalSignalByteSize = sizeof(float) * totalSignalLength;


	float* d_signalMatrix;

	hipMalloc(&d_signalMatrix, totalSignalByteSize);

	//Generate the signal!
	//Generate random numbers on the device
	//Generate random numbers using a normal distribution
	//Normal distribution should emulate white noise hopefully?
	//Generate signal
	if(hiprandGenerateNormal(*rngGen, d_signalMatrix, totalSignalLength, mean, stdDev) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Error when generating the signal\n");
		exit(1);
	}


	hipDeviceSynchronize();

	return d_signalMatrix;
}
*/



hipComplex* Device_GenerateWhiteNoiseSignalComplex(hiprandGenerator_t* rngGen, uint64_t h_valuesPerSample, uint64_t h_numberOfSamples,
		uint64_t h_batchSize, uint64_t h_threadNum)
{
	uint64_t totalSignalLength = h_valuesPerSample * h_numberOfSamples * h_batchSize * h_threadNum;
	uint64_t totalSignalByteSize = sizeof(hipComplex) * totalSignalLength;


	hipComplex* d_signalMatrix;

	hipMalloc(&d_signalMatrix, totalSignalByteSize);

	//Generate the signal!
	//Generate random numbers on the device
	//Generate random numbers using a normal distribution
	//Normal distribution should emulate white noise hopefully?
	//Generate signal
	if(hiprandGenerateNormal(*rngGen, (float*)d_signalMatrix, totalSignalLength * 2, 0.0f, 1.0f) != HIPRAND_STATUS_SUCCESS) //* 2 for complex numbers
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignalComplex: Error when generating the signal\n");
		exit(1);
	}


	hipDeviceSynchronize();

	return d_signalMatrix;
}




void Device_CalculateMeanMatrices(RFIMMemoryStruct* RFIMStruct, float* d_signalMatrices)
{


	//Calculate d_meanVec
	//d_meanVec = d_oneMatrix (1 x h_numberOfSamples) * d_signal (transposed) (h_numberOfSamples x h_valuesPerSample ) matrix = 1 * h_valuesPerSample matrix
	//This each of the beams added up. It adds up the columns of transposed d_signal
	//---------------------------
	hipblasStatus_t cublasError;


	float alpha = 1.0f / RFIMStruct->h_numberOfSamples;
	float beta = 0;

	uint64_t signalMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;
	uint64_t meanVecOffset = RFIMStruct->h_valuesPerSample;

	uint64_t streamIndex = 0;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Set the cuda stream
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[streamIndex]);

		//Compute the mean vector
		//We use the same d_onevec each time
		cublasError = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
									1, RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples,
									&alpha, RFIMStruct->d_oneVec, 1,
									d_signalMatrices + (i * signalMatrixOffset), RFIMStruct->h_valuesPerSample, &beta,
									RFIMStruct->d_meanVec + (i * meanVecOffset), 1);


		//Check for errors
		if(cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_CalculateMeanMatrices: An error occured while computing d_meanVec\n");
			exit(1);
		}

		//Iterate stream index
		streamIndex += 1;
		if(streamIndex >= RFIMStruct->h_cudaStreamsLength)
		{
			streamIndex = 0;
		}


		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		cublasError = cublasGetError();

		if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "CalculateMeanMatrix 1 error\n");
		}
		*/

	}





	//Calculate mean matrix
	//mean matrix = outer product of the transposed d_meanVec with itself
	//d_meanMatrix = d_meanVec_Transposed (h_valuesPerSample x 1) * d_meanVec (1 x h_valuesPerSample)
	//--------------------------------------

	alpha = 1.0f;
	streamIndex = 0;

	uint64_t covarianceMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_valuesPerSample;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Set the cuda stream
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[streamIndex]);

		//Compute the mean outer product
		cublasError = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample, 1,
				&alpha, RFIMStruct->d_meanVec + (i * meanVecOffset), 1,
				RFIMStruct->d_meanVec + (i * meanVecOffset), 1, &beta,
				RFIMStruct->d_covarianceMatrix + (i * covarianceMatrixOffset), RFIMStruct->h_valuesPerSample);



		//Check for errors
		if(cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_CalculateMeanMatrices: An error occured while computing d_meanVec\n");
			exit(1);
		}

		//Iterate stream index
		streamIndex += 1;
		if(streamIndex >= RFIMStruct->h_cudaStreamsLength)
		{
			streamIndex = 0;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		cublasError = cublasGetError();

		if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "CalculateMeanMatrix 2 error\n");
		}
		*/
	}


}




void Device_CalculateMeanMatricesBatched(RFIMMemoryStructBatched* RFIMStruct, float** d_signalMatrices)
{
	//Calculate d_meanVec
	//d_meanVec = d_oneMatrix (1 x h_numberOfSamples) * d_signal (transposed) (h_numberOfSamples x h_valuesPerSample ) matrix = 1 * h_valuesPerSample matrix
	//This each of the beams added up. It adds up the columns of transposed d_signal
	//---------------------------
	hipblasStatus_t cublasError;


	float alpha = 1.0f / RFIMStruct->h_numberOfSamples;
	float beta = 0;

	//uint64_t signalMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;
	//uint64_t meanVecOffset = RFIMStruct->h_valuesPerSample;

	//uint64_t streamIndex = 0;



	//Set the cuda stream
	//hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[streamIndex]);

	//Compute the mean vector
	//We use the same d_onevec each time
	cublasError = hipblasSgemmBatched(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
								1, RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples,
								&alpha, (const float**)RFIMStruct->d_oneVecBatched, 1,
								(const float**)d_signalMatrices, RFIMStruct->h_valuesPerSample, &beta,
								RFIMStruct->d_meanVecBatched, 1,
								RFIMStruct->h_batchSize);


	//Check for errors
	if(cublasError != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_CalculateMeanMatrices: An error occured while computing d_meanVec\n");
		exit(1);
	}


	/*
	//TODO: DEBUG REMOVE
	hipError_t hipError_t = hipDeviceSynchronize();
	cublasError = cublasGetError();

	if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "CalculateMeanMatrix 1 error\n");
	}
	*/



	//Calculate mean matrix
	//mean matrix = outer product of the transposed d_meanVec with itself
	//d_meanMatrix = d_meanVec_Transposed (h_valuesPerSample x 1) * d_meanVec (1 x h_valuesPerSample)
	//--------------------------------------

	alpha = 1.0f;

	//Compute the mean outer product
	cublasError = hipblasSgemmBatched(*RFIMStruct->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample, 1,
			&alpha, (const float**)RFIMStruct->d_meanVecBatched, 1,
			(const float**)RFIMStruct->d_meanVecBatched, 1, &beta,
			RFIMStruct->d_covarianceMatrixBatched, RFIMStruct->h_valuesPerSample,
			RFIMStruct->h_batchSize);



	//Check for errors
	if(cublasError != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_CalculateMeanMatrices: An error occured while computing d_meanVec\n");
		exit(1);
	}


}






void Device_CalculateMeanMatricesComplex(RFIMMemoryStructComplex* RFIMStruct, hipComplex* d_signalMatrices)
{
	//Calculate d_meanVec
	//d_meanVec = d_oneMatrix (1 x h_numberOfSamples) * d_signal (transposed) (h_numberOfSamples x h_valuesPerSample ) matrix = 1 * h_valuesPerSample matrix
	//This each of the beams added up. It adds up the columns of transposed d_signal
	//---------------------------
	hipblasStatus_t cublasError;


	hipComplex alpha = make_hipComplex(1.0f / RFIMStruct->h_numberOfSamples, 0) ;
	hipComplex beta = make_hipComplex(0, 0);

	uint64_t signalMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;
	uint64_t meanVecOffset = RFIMStruct->h_valuesPerSample;

	uint64_t streamIndex = 0;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Set the cuda stream
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[streamIndex]);

		//Compute the mean vector
		//We use the same d_onevec each time
		cublasError = hipblasCgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_C,
									1, RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples,
									&alpha, RFIMStruct->d_oneVec, 1,
									d_signalMatrices + (i * signalMatrixOffset), RFIMStruct->h_valuesPerSample, &beta,
									RFIMStruct->d_meanVec + (i * meanVecOffset), 1);


		//Check for errors
		if(cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_CalculateMeanMatricesComplex: An error occured while computing d_meanVec\n");
			exit(1);
		}

		//Iterate stream index
		streamIndex += 1;
		if(streamIndex >= RFIMStruct->h_cudaStreamsLength)
		{
			streamIndex = 0;
		}


		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		cublasError = cublasGetError();

		if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "CalculateMeanMatrix 1 error\n");
		}
		*/

	}





	//Calculate mean matrix
	//mean matrix = outer product of the transposed d_meanVec with itself
	//d_meanMatrix = d_meanVec_Transposed (h_valuesPerSample x 1) * d_meanVec (1 x h_valuesPerSample)
	//--------------------------------------

	alpha = make_hipComplex(1.0f, 0);
	streamIndex = 0;

	uint64_t covarianceMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_valuesPerSample;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Set the cuda stream
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[streamIndex]);

		//Compute the mean outer product
		cublasError = hipblasCgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_C, HIPBLAS_OP_N,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample, 1,
				&alpha, RFIMStruct->d_meanVec + (i * meanVecOffset), 1,
				RFIMStruct->d_meanVec + (i * meanVecOffset), 1, &beta,
				RFIMStruct->d_covarianceMatrix + (i * covarianceMatrixOffset), RFIMStruct->h_valuesPerSample);



		//Check for errors
		if(cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_CalculateMeanMatricesComplex: An error occured while computing d_meanVec\n");
			exit(1);
		}

		//Iterate stream index
		streamIndex += 1;
		if(streamIndex >= RFIMStruct->h_cudaStreamsLength)
		{
			streamIndex = 0;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		cublasError = cublasGetError();

		if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "CalculateMeanMatrix 2 error\n");
		}
		*/
	}
}



void Host_CalculateMeanMatrices(RFIMMemoryStructCPU* RFIMStruct, float* h_signalMatrices)
{
	float alphaOne = 1.0f / RFIMStruct->h_numberOfSamples;
	float alphaTwo = 1;
	float beta = 0;

	uint64_t signalMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;
	uint64_t meanVecOffset = RFIMStruct->h_valuesPerSample;
	uint64_t covarianceMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_valuesPerSample;


	//Compute the mean vector
	//We use the same d_onevec each time
	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Calculate meanVec
		cblas_sgemm(CblasColMajor, CblasNoTrans, CblasTrans,
				1, RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples,
				alphaOne, RFIMStruct->h_oneVec, 1,
				h_signalMatrices + (i * signalMatrixOffset), RFIMStruct->h_valuesPerSample, beta,
				RFIMStruct->h_meanVec + (i * meanVecOffset), 1);


		//Calculate mean outer product matrix
		cblas_sgemm(CblasColMajor, CblasTrans, CblasNoTrans,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample, 1,
				alphaTwo, RFIMStruct->h_meanVec + (i * meanVecOffset), 1,
				RFIMStruct->h_meanVec + (i * meanVecOffset), 1, beta,
				RFIMStruct->h_covarianceMatrix + (i * covarianceMatrixOffset), RFIMStruct->h_valuesPerSample);

	}



}






void Device_CalculateCovarianceMatrix(RFIMMemoryStruct* RFIMStruct, float* d_signalMatrices)
{

	//d_signalMatrix should be column-major as CUBLAS is column-major library (indexes start at 1 also)
	//Remember to take that into account!


	//Calculate the meanMatrix of the signal
	//--------------------------------

	Device_CalculateMeanMatrices(RFIMStruct, d_signalMatrices);

	//--------------------------------



	//Calculate the covariance matrix
	//-------------------------------
	//1. Calculate the outer product of the signal (sampleElements x sampleNumber) * ( sampleNumber x sampleElements)
	//	AKA. signal * (signal)T, where T = transpose, which will give you a (sampleNumber x sampleNumber) matrix as a result

	//Take the outer product of the signal with itself
	float alpha = 1.0f / RFIMStruct->h_numberOfSamples;
	float beta = -1;

	uint64_t signalOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;
	uint64_t covarianceMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_valuesPerSample;

	uint64_t cudaStreamIterator = 0;

	hipblasStatus_t cublasError;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		//Set the stream for the library
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[cudaStreamIterator]);

		cublasError = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples,
				&alpha, d_signalMatrices + (i * signalOffset), RFIMStruct->h_valuesPerSample,
				d_signalMatrices + (i * signalOffset), RFIMStruct->h_valuesPerSample, &beta,
				RFIMStruct->d_covarianceMatrix + (i * covarianceMatrixOffset), RFIMStruct->h_valuesPerSample);


		if(cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_CalculateCovarianceMatrix: error calculating the covariance matrix\n");
			exit(1);
		}


		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		cublasError = cublasGetError();

		if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_CalculateCovarianceMatrix 1 error\n");
		}
		*/

	}


}




void Device_CalculateCovarianceMatrixBatched(RFIMMemoryStructBatched* RFIMStruct, float** d_signalMatrices)
{
	//d_signalMatrix should be column-major as CUBLAS is column-major library (indexes start at 1 also)
	//Remember to take that into account!


	//Calculate the meanMatrix of the signal
	//--------------------------------

	//Set the stream to stream zero, so it's not on the default stream
	hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[0]);

	Device_CalculateMeanMatricesBatched(RFIMStruct, d_signalMatrices);

	//--------------------------------



	//Calculate the covariance matrix
	//-------------------------------
	//1. Calculate the outer product of the signal (sampleElements x sampleNumber) * ( sampleNumber x sampleElements)
	//	AKA. signal * (signal)T, where T = transpose, which will give you a (sampleNumber x sampleNumber) matrix as a result

	//Take the outer product of the signal with itself
	float alpha = 1.0f / RFIMStruct->h_numberOfSamples;
	float beta = -1;

	hipblasStatus_t cublasError;


	cublasError = hipblasSgemmBatched(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
			RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples,
			&alpha, (const float**)d_signalMatrices, RFIMStruct->h_valuesPerSample,
			(const float**)d_signalMatrices, RFIMStruct->h_valuesPerSample, &beta,
			RFIMStruct->d_covarianceMatrixBatched, RFIMStruct->h_valuesPerSample,
			RFIMStruct->h_batchSize);


	if(cublasError != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_CalculateCovarianceMatrix: error calculating the covariance matrix\n");
		exit(1);
	}



	/*
	//TODO: DEBUG REMOVE
	hipError_t hipError_t = hipDeviceSynchronize();
	cublasError = cublasGetError();

	if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_CalculateCovarianceMatrix 1 error\n");
	}
	*/


}







void Device_CalculateCovarianceMatrixComplex(RFIMMemoryStructComplex* RFIMStruct, hipComplex* d_signalMatrices)
{
	//d_signalMatrix should be column-major as CUBLAS is column-major library (indexes start at 1 also)
	//Remember to take that into account!


	//Calculate the meanMatrix of the signal
	//--------------------------------

	Device_CalculateMeanMatricesComplex(RFIMStruct, d_signalMatrices);

	//--------------------------------



	//Calculate the covariance matrix
	//-------------------------------
	//1. Calculate the outer product of the signal (sampleElements x sampleNumber) * ( sampleNumber x sampleElements)
	//	AKA. signal * (signal)T, where T = transpose, which will give you a (sampleNumber x sampleNumber) matrix as a result

	//Take the outer product of the signal with itself
	hipComplex alpha = make_hipComplex(1.0f / RFIMStruct->h_numberOfSamples, 0);
	hipComplex beta = make_hipComplex(-1, 0);

	uint64_t signalOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;
	uint64_t covarianceMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_valuesPerSample;

	uint64_t cudaStreamIterator = 0;

	hipblasStatus_t cublasError;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		//Set the stream for the library
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[cudaStreamIterator]);

		cublasError = hipblasCgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_C,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples,
				&alpha, d_signalMatrices + (i * signalOffset), RFIMStruct->h_valuesPerSample,
				d_signalMatrices + (i * signalOffset), RFIMStruct->h_valuesPerSample, &beta,
				RFIMStruct->d_covarianceMatrix + (i * covarianceMatrixOffset), RFIMStruct->h_valuesPerSample);


		if(cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_CalculateCovarianceMatrixComplex: error calculating the covariance matrix\n");
			exit(1);
		}


		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		cublasError = cublasGetError();

		if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_CalculateCovarianceMatrix 1 error\n");
		}
		*/

	}

}




void Host_CalculateCovarianceMatrix(RFIMMemoryStructCPU* RFIMStruct, float* signalMatrices)
{
	//Calculate the mean matrices
	Host_CalculateMeanMatrices(RFIMStruct, signalMatrices);


	//Calculate the covariance matrices
	//Take the outer product of the signal with itself
	float alpha = 1.0f / RFIMStruct->h_numberOfSamples;
	float beta = -1;

	uint64_t signalOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;
	uint64_t covarianceMatrixOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_valuesPerSample;




	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		cblas_sgemm(CblasColMajor, CblasNoTrans, CblasTrans,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples,
				alpha, signalMatrices + (i * signalOffset), RFIMStruct->h_valuesPerSample,
				signalMatrices + (i * signalOffset), RFIMStruct->h_valuesPerSample, beta,
				RFIMStruct->h_covarianceMatrix + (i * covarianceMatrixOffset), RFIMStruct->h_valuesPerSample);

	}

}





void Device_EigenvalueSolver(RFIMMemoryStruct* RFIMStruct)
{

	hipsolverStatus_t cusolverStatus;

	uint64_t cudaStreamIterator = 0;


	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Set the stream
		hipsolverSetStream(*RFIMStruct->cusolverHandle, RFIMStruct->h_cudaStreams[cudaStreamIterator]);

		//Tell the device to solve the eigenvectors
		cusolverStatus = hipsolverDnSgesvd(*RFIMStruct->cusolverHandle, 'A', 'A',
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_covarianceMatrix + (i * RFIMStruct->h_covarianceMatrixBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_S + (i * RFIMStruct->h_SBatchOffset),
				RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_VT + (i * RFIMStruct->h_VTBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_eigenWorkingSpace + (i * RFIMStruct->h_eigenWorkingSpaceBatchOffset),
				RFIMStruct->h_singleEigWorkingSpaceByteSize,
				NULL,
				RFIMStruct->d_devInfo + (i * RFIMStruct->h_devInfoBatchOffset));


		//Check for startup errors
		if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
		{

			if(cusolverStatus == HIPSOLVER_STATUS_NOT_INITIALIZED)
				printf("1\n");
			if(cusolverStatus == HIPSOLVER_STATUS_INVALID_VALUE)
				printf("2\n");
			if(cusolverStatus == HIPSOLVER_STATUS_ARCH_MISMATCH)
				printf("3\n");
			if(cusolverStatus == HIPSOLVER_STATUS_INTERNAL_ERROR)
				printf("4\n");


			fprintf(stderr, "Device_EigenvalueSolver: Error solving eigenvalues\n");
			exit(1);

		}


		//Put in a request to copy the devInfo back to the host so you can check it later
		hipMemcpyAsync(RFIMStruct->h_devInfo + (i * RFIMStruct->h_devInfoBatchOffset),
				RFIMStruct->d_devInfo + (i * RFIMStruct->h_devInfoBatchOffset), sizeof(int),
				hipMemcpyDeviceToHost, RFIMStruct->h_cudaStreams[cudaStreamIterator]);



		//Iterate to the next stream
		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		hipblasStatus_t cublasError = cublasGetError();

		if(hipError_t != hipSuccess )
		{
			fprintf(stderr, "Device_EigenvalueSolver 1 error\n");
		}
		*/

	}



	/*
	//TODO: ****************** EXPERIMENT WITH PUTTING THIS AT THE END OF THE RFIM ROUTINE ******************
	//Wait for everything to complete
	for(uint64_t i = 0; i < RFIMStruct->h_cudaStreamsLength; ++i)
	{
		hipStreamSynchronize(RFIMStruct->h_cudaStreams[i]);
	}


	//Check each devInfo value
	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		if(RFIMStruct->h_devInfo[i] != 0)
		{
			fprintf(stderr, "Device_EigenvalueSolver: Error with the %dth parameter on the %lluth batch\n", RFIMStruct->h_devInfo[i], i);
			exit(1);
		}
	}

*/

	//********************************************************************************************************

	/*
	//TODO: DEBUG REMOVE
	hipError_t hipError_t = hipDeviceSynchronize();


	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "Device_EigenvalueSolver 2 error\n");
	}
	*/

}




void Device_EigenvalueSolverBatched(RFIMMemoryStructBatched* RFIMStruct)
{


	//Have to wait for all batched calls to finish
	hipStreamSynchronize(RFIMStruct->h_cudaStreams[0]);


	hipsolverStatus_t cusolverStatus;

	//use streams 1 or greater for memcopies and eigenvalue solving
	uint64_t cudaStreamIterator = 1;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Set the stream
		hipsolverSetStream(*RFIMStruct->cusolverHandle, RFIMStruct->h_cudaStreams[cudaStreamIterator]);

		//Tell the device to solve the eigenvectors
		cusolverStatus = hipsolverDnSgesvd(*RFIMStruct->cusolverHandle, 'A', 'A',
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_covarianceMatrix + (i * RFIMStruct->h_covarianceMatrixBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_S + (i * RFIMStruct->h_SBatchOffset),
				RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_VT + (i * RFIMStruct->h_VTBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_eigenWorkingSpace + (i * RFIMStruct->h_eigenWorkingSpaceBatchOffset),
				RFIMStruct->h_singleEigWorkingSpaceByteSize,
				NULL,
				RFIMStruct->d_devInfo + (i * RFIMStruct->h_devInfoBatchOffset));


		//Check for startup errors
		if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
		{

			if(cusolverStatus == HIPSOLVER_STATUS_NOT_INITIALIZED)
				printf("1\n");
			if(cusolverStatus == HIPSOLVER_STATUS_INVALID_VALUE)
				printf("2\n");
			if(cusolverStatus == HIPSOLVER_STATUS_ARCH_MISMATCH)
				printf("3\n");
			if(cusolverStatus == HIPSOLVER_STATUS_INTERNAL_ERROR)
				printf("4\n");


			fprintf(stderr, "Device_EigenvalueSolver: Error solving eigenvalues\n");
			exit(1);

		}


		//Put in a request to copy the devInfo back to the host so you can check it later
		hipMemcpyAsync(RFIMStruct->h_devInfo + (i * RFIMStruct->h_devInfoBatchOffset),
				RFIMStruct->d_devInfo + (i * RFIMStruct->h_devInfoBatchOffset), sizeof(int),
				hipMemcpyDeviceToHost, RFIMStruct->h_cudaStreams[cudaStreamIterator]);



		//Iterate to the next stream
		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			//use streams 1 or greater for memcopies and eigenvalue solving
			cudaStreamIterator = 1;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		hipblasStatus_t cublasError = cublasGetError();

		if(hipError_t != hipSuccess )
		{
			fprintf(stderr, "Device_EigenvalueSolver 1 error\n");
		}
		*/

	}



	/*
	//TODO: ****************** EXPERIMENT WITH PUTTING THIS AT THE END OF THE RFIM ROUTINE ******************
	//Wait for everything to complete
	for(uint64_t i = 0; i < RFIMStruct->h_cudaStreamsLength; ++i)
	{
		hipStreamSynchronize(RFIMStruct->h_cudaStreams[i]);
	}


	//Check each devInfo value
	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		if(RFIMStruct->h_devInfo[i] != 0)
		{
			fprintf(stderr, "Device_EigenvalueSolver: Error with the %dth parameter on the %lluth batch\n", RFIMStruct->h_devInfo[i], i);
			exit(1);
		}
	}

*/

	//********************************************************************************************************

	/*
	//TODO: DEBUG REMOVE
	hipError_t hipError_t = hipDeviceSynchronize();


	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "Device_EigenvalueSolver 2 error\n");
	}
	*/
}







void Device_EigenvalueSolverComplex(RFIMMemoryStructComplex* RFIMStruct)
{
	hipsolverStatus_t cusolverStatus;

	uint64_t cudaStreamIterator = 0;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Set the stream
		hipsolverSetStream(*RFIMStruct->cusolverHandle, RFIMStruct->h_cudaStreams[cudaStreamIterator]);

		//Tell the device to solve the eigenvectors
		cusolverStatus = hipsolverDnCgesvd(*RFIMStruct->cusolverHandle, 'A', 'A',
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_covarianceMatrix + (i * RFIMStruct->h_covarianceMatrixBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_S + (i * RFIMStruct->h_SBatchOffset),
				RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_VT + (i * RFIMStruct->h_VTBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_eigenWorkingSpace + (i * RFIMStruct->h_eigenWorkingSpaceBatchOffset),
				RFIMStruct->h_singleEigWorkingSpaceByteSize,
				RFIMStruct->h_rWork + (i * RFIMStruct->h_rWorkBatchOffset),
				RFIMStruct->d_devInfo + (i * RFIMStruct->h_devInfoBatchOffset));


		//Check for startup errors
		if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
		{

			if(cusolverStatus == HIPSOLVER_STATUS_NOT_INITIALIZED)
				printf("1\n");
			if(cusolverStatus == HIPSOLVER_STATUS_INVALID_VALUE)
				printf("2\n");
			if(cusolverStatus == HIPSOLVER_STATUS_ARCH_MISMATCH)
				printf("3\n");
			if(cusolverStatus == HIPSOLVER_STATUS_INTERNAL_ERROR)
				printf("4\n");


			fprintf(stderr, "Device_EigenvalueSolver: Error solving eigenvalues\n");
			exit(1);

		}


		//Put in a request to copy the devInfo back to the host so you can check it later
		hipMemcpyAsync(RFIMStruct->h_devInfo + (i * RFIMStruct->h_devInfoBatchOffset),
				RFIMStruct->d_devInfo + (i * RFIMStruct->h_devInfoBatchOffset), sizeof(int),
				hipMemcpyDeviceToHost, RFIMStruct->h_cudaStreams[cudaStreamIterator]);



		//Iterate to the next stream
		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		hipblasStatus_t cublasError = cublasGetError();

		if(hipError_t != hipSuccess )
		{
			fprintf(stderr, "Device_EigenvalueSolver 1 error\n");
		}
		*/

	}



	//TODO: ****************** EXPERIMENT WITH PUTTING THIS AT THE END OF THE RFIM ROUTINE ******************
	//Wait for everything to complete
	for(uint64_t i = 0; i < RFIMStruct->h_cudaStreamsLength; ++i)
	{
		hipStreamSynchronize(RFIMStruct->h_cudaStreams[i]);
	}


	//Check each devInfo value
	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		if(RFIMStruct->h_devInfo[i] != 0)
		{
			fprintf(stderr, "Device_EigenvalueSolver: Error with the %dth parameter on the %lluth batch\n", RFIMStruct->h_devInfo[i], i);
			exit(1);
		}
	}


	//********************************************************************************************************

	/*
	//TODO: DEBUG REMOVE
	hipError_t hipError_t = hipDeviceSynchronize();


	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "Device_EigenvalueSolver 2 error\n");
	}
	*/
}




void Host_EigenvalueSolver(RFIMMemoryStructCPU* RFIMStruct)
{
	int info;

	//Compute the SVD for each covariance matrix
	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		info =  LAPACKE_sgesdd(LAPACK_COL_MAJOR, 'A',
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample,
				RFIMStruct->h_covarianceMatrix + (i * RFIMStruct->h_covarianceMatrixBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->h_S + (i * RFIMStruct->h_SBatchOffset),
				RFIMStruct->h_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->h_VT + (i * RFIMStruct->h_VTBatchOffset), RFIMStruct->h_valuesPerSample);


		//Check to see if everything went ok
		if(info != 0)
		{
			//If info = -i, the i-th parameter had an illegal value
			//If info = i, then sgesdd did not converge, updataing process failed
			fprintf(stderr, "Host_EigenvalueSolver: SVD computation didn't converge. Info: %d\n", info);
			exit(1);
		}

		/*
		//Tell the device to solve the eigenvectors
		cusolverStatus = hipsolverDnSgesvd(*RFIMStruct->cusolverHandle, 'A', 'A',
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_covarianceMatrix + (i * RFIMStruct->h_covarianceMatrixBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_S + (i * RFIMStruct->h_SBatchOffset),
				RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_VT + (i * RFIMStruct->h_VTBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_eigenWorkingSpace + (i * RFIMStruct->h_eigenWorkingSpaceBatchOffset),
				RFIMStruct->h_singleEigWorkingSpaceByteSize,
				NULL,
				RFIMStruct->d_devInfo + (i * RFIMStruct->h_devInfoBatchOffset));
	*/
	}

}




//Eigenvector reduction and signal projection/filtering
//All matrices are column-major

//h_eigenVectorDimensionsToReduce is the number of eigenvectors to remove from the eigenvector matrix, for now it's 2

//Os = Original signal matrix
//A column-major matrix containing the signal
//It has dimensionality: h_valuesPerSample * h_numberOfSamples, which will probably be 26 x 1024?

//Er = Reduced Eigenvector matrix.
//The eigenvectors of the fully symmetrical covariance matrix, with some of the eigenvectors removed.
//It has dimensions: h_valuesPerSample x (h_valuesPerSample - h_eigenVectorDimensionsToReduce), probably 26 x 24?

//Ps = Projected signal matrix.
//The original data projected along the reduced eigenvector axis's
//This matrix will have dimensions: (h_valuesPerSample - h_eigenVectorDimensionsToReduce) x h_numberOfSamples, probably 24 x 1024?

//Fs = Final signal matrix
//This is the original data projected into the lower reduced eigenvector dimensionality, then back into the original dimensionality. This has the effect of flattening data along the removed dimensions. It may add correlations were there was previously none?
//But should also hopefully remove some RFI
//It will have dimensions: h_valuesPerSample * h_numberOfSamples, probably 26 x 1024?


//Equations!
// Ps = (Er Transposed) * Os
// Fs = Er * Ps      Note that the inverse of Er should just be its transpose, even if you remove some of the eigenvectors. This is because all the eigenvectors are orthogonal unit vectors (or should be anyway...)


//Steps
//1. Remove RFIMStruct->h_eigenVectorDimensionsToReduce dimensions from the eigenvector matrix (this is done via pointer arithmetic rather than actually removing the data) THIS WON'T WORK IF THE COLUMNS TO REMOVE ARE NOT ALL NEXT TO EACH OTHER!
//2. Compute the matrix Ps
//3. Compute the matrix Fs (final signal matrix)
//4. Pass on Fs, down the line? Keep it on the GPU? Copy it to the host? Write it to a file in the file system? Dunno.



void Device_EigenReductionAndFiltering(RFIMMemoryStruct* RFIMStruct, float* d_originalSignalMatrices, float* d_filteredSignals)
{


	//Set the appropriate number of columns to zero
	uint64_t eigenvectorZeroByteSize = sizeof(float) * RFIMStruct->h_valuesPerSample * RFIMStruct->h_eigenVectorDimensionsToReduce;

	uint64_t cudaStreamIterator = 0;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		hipMemsetAsync(RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset),
				0, eigenvectorZeroByteSize, RFIMStruct->h_cudaStreams[cudaStreamIterator]);

		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		hipblasStatus_t cublasError = cublasGetError();

		if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_EigenReductionAndFiltering 1 error\n");
		}
		*/


	}



	hipblasStatus_t hipblasStatus_t;

	//Projected signal matrix
	//Ps = (Er Transposed) * Os
	float alpha = 1;
	float beta = 0;

	uint64_t originalSignalBatchOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;

	cudaStreamIterator = 0;


	//Do the projection
	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		//Set the stream
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[cudaStreamIterator]);



		//compute
		hipblasStatus_t = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
				&alpha, RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				d_originalSignalMatrices + (i * originalSignalBatchOffset), RFIMStruct->h_valuesPerSample, &beta,
				RFIMStruct->d_projectedSignalMatrix + (i * RFIMStruct->h_projectedSignalBatchOffset), RFIMStruct->h_valuesPerSample);


		//Check request status codes
		if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_EigenReductionAndFiltering: error calculating the projected signal\n");
			exit(1);
		}


		//Iterate the stream
		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();

		hipblasStatus_t = cublasGetError();

		if(hipError_t != hipSuccess || hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_EigenReductionAndFiltering 2 error\n");
		}
		*/
	}



	//Do the reprojection back
	//final signal matrix
	// Fs = Er * Ps

	cudaStreamIterator = 0;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Set the stream
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[cudaStreamIterator]);


		hipblasStatus_t = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
				&alpha, RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_projectedSignalMatrix + (i * RFIMStruct->h_projectedSignalBatchOffset), RFIMStruct->h_valuesPerSample, &beta,
				d_filteredSignals + (i * originalSignalBatchOffset), RFIMStruct->h_valuesPerSample);




		if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_EigenReductionAndFiltering: error calculating the filtered signal\n");
			exit(1);
		}


		//Iterate the stream
		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		hipblasStatus_t = cublasGetError();

		if(hipError_t != hipSuccess || hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_EigenReductionAndFiltering 3 error\n");
		}
		*/
	}



}




void Device_EigenReductionAndFilteringBatched(RFIMMemoryStructBatched* RFIMStruct, float** d_originalSignalMatrices, float** d_filteredSignals)
{
	//Set the appropriate number of columns to zero
	uint64_t eigenvectorZeroByteSize = sizeof(float) * RFIMStruct->h_valuesPerSample * RFIMStruct->h_eigenVectorDimensionsToReduce;

	//use streams 1 or greater for memcopies and eigenvalue solving
	uint64_t cudaStreamIterator = 1;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		hipMemsetAsync(RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset),
				0, eigenvectorZeroByteSize, RFIMStruct->h_cudaStreams[cudaStreamIterator]);

		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 1;
		}

		/*
		//TODO: DEBUG REMOVE
		hipError_t hipError_t = hipDeviceSynchronize();
		hipblasStatus_t cublasError = cublasGetError();

		if(hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_EigenReductionAndFiltering 1 error\n");
		}
		*/
	}


	//Wait for all the memsets to complete before stating the compute
	for(uint64_t i = 1; i < RFIMStruct->h_cudaStreamsLength; ++i)
	{
		hipStreamSynchronize(RFIMStruct->h_cudaStreams[i]);
	}



	hipblasStatus_t hipblasStatus_t;

	//Projected signal matrix
	//Ps = (Er Transposed) * Os
	float alpha = 1;
	float beta = 0;


	//Do the projection
	//compute

	//Set the stream to stream zero, so it's not on the default stream
	hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[0]);

	hipblasStatus_t = hipblasSgemmBatched(*RFIMStruct->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
			&alpha, (const float**)RFIMStruct->d_UBatched, RFIMStruct->h_valuesPerSample,
			(const float**)d_originalSignalMatrices, RFIMStruct->h_valuesPerSample, &beta,
			RFIMStruct->d_projectedSignalMatrixBatched, RFIMStruct->h_valuesPerSample,
			RFIMStruct->h_batchSize);


	//Check request status codes
	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_EigenReductionAndFiltering: error calculating the projected signal\n");
		exit(1);
	}


	/*
	//TODO: DEBUG REMOVE
	hipError_t hipError_t = hipDeviceSynchronize();

	hipblasStatus_t = cublasGetError();

	if(hipError_t != hipSuccess || hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_EigenReductionAndFiltering 2 error\n");
	}
	*/



	//Do the reprojection back
	//final signal matrix
	// Fs = Er * Ps



	hipblasStatus_t = hipblasSgemmBatched(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
			&alpha, (const float**)RFIMStruct->d_UBatched, RFIMStruct->h_valuesPerSample,
			(const float**)RFIMStruct->d_projectedSignalMatrixBatched, RFIMStruct->h_valuesPerSample, &beta,
			d_filteredSignals, RFIMStruct->h_valuesPerSample,
			RFIMStruct->h_batchSize);




	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_EigenReductionAndFiltering: error calculating the filtered signal\n");
		exit(1);
	}

	/*
	//TODO: DEBUG REMOVE
	hipError_t hipError_t = hipDeviceSynchronize();
	hipblasStatus_t = cublasGetError();

	if(hipError_t != hipSuccess || hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_EigenReductionAndFiltering 3 error\n");
	}
	*/

}




void Device_EigenReductionAndFilteringComplex(RFIMMemoryStructComplex* RFIMStruct, hipComplex* d_originalSignalMatrices, hipComplex* d_filteredSignals)
{
	//Set the appropriate number of columns to zero
	uint64_t eigenvectorZeroByteSize = sizeof(hipComplex) * RFIMStruct->h_valuesPerSample * RFIMStruct->h_eigenVectorDimensionsToReduce;

	uint64_t cudaStreamIterator = 0;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		hipMemsetAsync(RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset),
				0, eigenvectorZeroByteSize, RFIMStruct->h_cudaStreams[cudaStreamIterator]);

		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

	}


	hipblasStatus_t hipblasStatus_t;

	//Projected signal matrix
	//Ps = (Er Transposed) * Os
	hipComplex alpha = make_hipComplex(1, 0);
	hipComplex beta = make_hipComplex(0, 0);

	uint64_t originalSignalBatchOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;

	cudaStreamIterator = 0;


	//Do the projection
	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		//Set the stream
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[cudaStreamIterator]);



		//compute
		hipblasStatus_t = hipblasCgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_C, HIPBLAS_OP_N,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
				&alpha, RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				d_originalSignalMatrices + (i * originalSignalBatchOffset), RFIMStruct->h_valuesPerSample, &beta,
				RFIMStruct->d_projectedSignalMatrix + (i * RFIMStruct->h_projectedSignalBatchOffset), RFIMStruct->h_valuesPerSample);


		//Check request status codes
		if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_EigenReductionAndFiltering: error calculating the projected signal\n");
			exit(1);
		}


		//Iterate the stream
		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

	}



	//Do the reprojection back
	//final signal matrix
	// Fs = Er * Ps

	cudaStreamIterator = 0;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Set the stream
		hipblasSetStream(*RFIMStruct->cublasHandle, RFIMStruct->h_cudaStreams[cudaStreamIterator]);


		hipblasStatus_t = hipblasCgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
				&alpha, RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_projectedSignalMatrix + (i * RFIMStruct->h_projectedSignalBatchOffset), RFIMStruct->h_valuesPerSample, &beta,
				d_filteredSignals + (i * originalSignalBatchOffset), RFIMStruct->h_valuesPerSample);




		if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "Device_EigenReductionAndFiltering: error calculating the filtered signal\n");
			exit(1);
		}


		//Iterate the stream
		cudaStreamIterator += 1;
		if(cudaStreamIterator >= RFIMStruct->h_cudaStreamsLength)
		{
			cudaStreamIterator = 0;
		}

	}



}



void Host_EigenReductionAndFiltering(RFIMMemoryStructCPU* RFIMStruct, float* h_originalSignalMatrices, float* h_filtredSignalMatrices)
{

	//Set the appropriate number of columns to zero
	uint64_t eigenvectorZeroByteSize = sizeof(float) * RFIMStruct->h_valuesPerSample * RFIMStruct->h_eigenVectorDimensionsToReduce;

	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{
		memset(RFIMStruct->h_U + (i * RFIMStruct->h_UBatchOffset), 0, eigenvectorZeroByteSize);

		/*
		hipMemsetAsync(RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset),
				0, eigenvectorZeroByteSize, RFIMStruct->h_cudaStreams[cudaStreamIterator]);
		*/
	}




	float alpha = 1;
	float beta = 0;

	uint64_t originalSignalBatchOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_numberOfSamples;


	//Do the projection
	for(uint64_t i = 0; i < RFIMStruct->h_batchSize; ++i)
	{

		//Projected signal matrix
		//Ps = (Er Transposed) * Os
		cblas_sgemm(CblasColMajor, CblasTrans, CblasNoTrans,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
				alpha, RFIMStruct->h_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				h_originalSignalMatrices + (i * originalSignalBatchOffset), RFIMStruct->h_valuesPerSample, beta,
				RFIMStruct->h_projectedSignalMatrix + (i * RFIMStruct->h_projectedSignalBatchOffset), RFIMStruct->h_valuesPerSample);


		/*
		hipblasStatus_t = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
				&alpha, RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				d_originalSignalMatrices + (i * originalSignalBatchOffset), RFIMStruct->h_valuesPerSample, &beta,
				RFIMStruct->d_projectedSignalMatrix + (i * RFIMStruct->h_projectedSignalBatchOffset), RFIMStruct->h_valuesPerSample);

		*/


		//Do the reprojection back
		//final signal matrix
		// Fs = Er * Ps

		cblas_sgemm(CblasColMajor, CblasNoTrans, CblasNoTrans,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
				alpha,  RFIMStruct->h_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->h_projectedSignalMatrix + (i * RFIMStruct->h_projectedSignalBatchOffset), RFIMStruct->h_valuesPerSample, beta,
				h_filtredSignalMatrices + (i * originalSignalBatchOffset), RFIMStruct->h_valuesPerSample);


		/*
		hipblasStatus_t = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
				&alpha, RFIMStruct->d_U + (i * RFIMStruct->h_UBatchOffset), RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_projectedSignalMatrix + (i * RFIMStruct->h_projectedSignalBatchOffset), RFIMStruct->h_valuesPerSample, &beta,
				d_filteredSignals + (i * originalSignalBatchOffset), RFIMStruct->h_valuesPerSample);

		*/
	}





}




/*
void Device_MatrixTranspose(hipblasHandle_t* cublasHandle, const float* d_matrix, float* d_matrixTransposed, uint64_t rowNum, uint64_t colNum)
{

	hipblasStatus_t hipblasStatus_t;

	float alpha = 1;
	float beta = 0;


	hipblasStatus_t = hipblasSgeam(*cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, colNum, rowNum,
			&alpha, d_matrix, rowNum,
			&beta, d_matrix, rowNum,
			d_matrixTransposed, colNum);


	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_InplaceMatrixTranspose: Transposition of the matrix failed!\n");
		//exit(1);
	}

}


*/


