#include "hip/hip_runtime.h"
/*
 * RFIMHelperFunctions.cu
 *
 *  Created on: 17/03/2016
 *      Author: vincentvillani
 */

#include "../Header/RFIMHelperFunctions.h"

#include <stdio.h>



#include "../Header/CudaUtilityFunctions.h"
#include "../Header/Kernels.h"
#include "../Header/CudaMacros.h"


//Private helper functions
//--------------------------





//Private functions implementation
//----------------------------------




//--------------------------

/*
float* Device_GenerateWhiteNoiseSignal(hiprandGenerator_t* rngGen, uint64_t h_valuesPerSample, uint64_t h_numberOfSamples)
{

	uint64_t totalSignalLength = h_valuesPerSample * h_numberOfSamples;
	uint64_t totalSignalByteSize = totalSignalLength * sizeof(float);

	float* d_signal;

	hipError_t error;

	//Allocate the memory required to store the signal
	error =  hipMalloc(&d_signal, totalSignalByteSize);

	//Check that it was allocated successfully
	if(error != hipSuccess)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Unable to allocate %llu bytes of memory on the device\n", totalSignalByteSize);
		exit(1);
	}


	//Generate the signal!
	//Generate random numbers on the device
	//Generate random numbers using a normal distribution
	//Normal distribution should emulate white noise hopefully?
	//Generate signal
	if(hiprandGenerateNormal(*rngGen, d_signal, totalSignalLength, 0.0f, 1.0f) != HIPRAND_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_GenerateWhiteNoiseSignal: Error when generating the signal\n");
		exit(1);
	}


	//Return the generated signal that resides in DEVICE memory
	return d_signal;

}



void Device_CalculateMeanMatrix(RFIMMemoryStruct* RFIMStruct, const float* d_signalMatrix)
{

	//Calculate d_meanVec
	//d_meanVec = d_oneMatrix (1 x h_numberOfSamples) * d_signal (transposed) (h_numberOfSamples x h_valuesPerSample ) matrix = 1 * h_valuesPerSample matrix
	//This each of the beams added up. It adds up the columns of transposed d_signal
	//---------------------------
	hipblasStatus_t cublasError;


	float alpha = 1.0f / RFIMStruct->h_numberOfSamples;
	float beta = 0;

	cublasError = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 1, RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples,
			&alpha, RFIMStruct->d_oneVec, 1,
			d_signalMatrix, RFIMStruct->h_valuesPerSample, &beta,
			RFIMStruct->d_meanVec, 1);



	if(cublasError != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "CalculateMeanMatrix: An error occured while computing d_meanVec\n");
		exit(1);
	}

	//--------------------------------------


	//Calculate mean matrix
	//mean matrix = outer product of the transposed d_meanVec with itself
	//d_meanMatrix = d_meanVec_Transposed (h_valuesPerSample x 1) * d_meanVec (1 x h_valuesPerSample)
	//--------------------------------------

	alpha = 1.0f;

	cublasError = hipblasSsyrk(*RFIMStruct->cublasHandle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, RFIMStruct->h_valuesPerSample, 1,
			&alpha, RFIMStruct->d_meanVec, 1, &beta, RFIMStruct->d_upperTriangularCovarianceMatrix, RFIMStruct->h_valuesPerSample);

	if(cublasError != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "CalculateMeanMatrix: An error occured while computing d_meanMatrix\n");
		exit(1);
	}

}




void Device_CalculateCovarianceMatrix(RFIMMemoryStruct* RFIMStruct, float* d_signalMatrix)
{
	//d_signalMatrix should be column-major as CUBLAS is column-major library (indexes start at 1 also)
	//Remember to take that into account!


	//Calculate the meanMatrix of the signal
	//--------------------------------

	Device_CalculateMeanMatrix(RFIMStruct, d_signalMatrix);


	//--------------------------------



	//Calculate the covariance matrix
	//-------------------------------
	//1. Calculate the outer product of the signal (sampleElements x sampleNumber) * ( sampleNumber x sampleElements)
	//	AKA. signal * (signal)T, where T = transpose, which will give you a (sampleNumber x sampleNumber) matrix as a result

	//Take the outer product of the signal with itself
	float alpha = 1.0f / RFIMStruct->h_numberOfSamples;
	float beta = -1;

	hipblasStatus_t cublasError;



	//At this point RFIMStruct->d_upperTriangularCovarianceMatrix is actually the upper triangular mean matrix,
	//this is done to get better performance out of the cublas API
	cublasError = hipblasSsyrk(*RFIMStruct->cublasHandle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, RFIMStruct->h_valuesPerSample,
			RFIMStruct->h_numberOfSamples,
			&alpha, d_signalMatrix, RFIMStruct->h_valuesPerSample,
			&beta, RFIMStruct->d_upperTriangularCovarianceMatrix, RFIMStruct->h_valuesPerSample);

	if(cublasError != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_CalculateCovarianceMatrix: error calculating the covariance matrix\n");
		exit(1);
	}





	//Calculate the full symmetric covariance matrix
	//1. Transpose the covariance matrix
	Device_MatrixTranspose(RFIMStruct->cublasHandle, RFIMStruct->d_upperTriangularCovarianceMatrix, RFIMStruct->d_upperTriangularTransposedMatrix,
			RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample);

	//2. Set the transposed covariance matrix diagonal to zero
	dim3 blockDim(32);
	dim3 gridDim(1, ceilf(RFIMStruct->h_valuesPerSample / (float)32));
	setDiagonalToZero<<<gridDim, blockDim>>> (RFIMStruct->d_upperTriangularTransposedMatrix, RFIMStruct->h_valuesPerSample);

	hipDeviceSynchronize();
	hipError_t hipError_t = hipGetLastError();

	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "Device_CalculateCovarianceMatrix: error when starting the kernel setDiagonalToZero\n");
		fprintf(stderr, "Grid size: (%u, %u) Block size: (%u, %u)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);
		fprintf(stderr, "hipError_t: %s", hipGetErrorString(hipError_t));
		exit(1);
	}

	//3. Add the two matrices together

	//TODO: Look into whether or not I need to do this. This memory is reused each time around
	//hipMemset(RFIMStruct->d_fullSymmetricCovarianceMatrix, 0, sizeof(float) * RFIMStruct->h_valuesPerSample * RFIMStruct->h_valuesPerSample);

	alpha = 1.0f;
	beta = 1.0f;

	cublasError = hipblasSgeam(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample,
				&alpha, RFIMStruct->d_upperTriangularCovarianceMatrix, RFIMStruct->h_valuesPerSample,
				&beta, RFIMStruct->d_upperTriangularTransposedMatrix, RFIMStruct->h_valuesPerSample,
				RFIMStruct->d_fullSymmetricCovarianceMatrix, RFIMStruct->h_valuesPerSample);




	if(cublasError != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_CalculateCovarianceMatrix: hipblasSgeam call failed\n");
		exit(1);
	}


}




void Device_MatrixTranspose(hipblasHandle_t* cublasHandle, const float* d_matrix, float* d_matrixTransposed, uint64_t rowNum, uint64_t colNum)
{

	hipblasStatus_t hipblasStatus_t;

	float alpha = 1;
	float beta = 0;


	hipblasStatus_t = hipblasSgeam(*cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, colNum, rowNum,
			&alpha, d_matrix, rowNum,
			&beta, d_matrix, rowNum,
			d_matrixTransposed, colNum);


	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_InplaceMatrixTranspose: Transposition of the matrix failed!\n");
		//exit(1);
	}

}



/*
float* Device_FullSymmetricMatrix(hipblasHandle_t* cublasHandle, const float* d_triangularMatrix, uint64_t rowAndColNum)
{
	float* d_fullMatrix;

	//Transpose the d_triangularMatrix
	//Transpose the matrix
	float* d_triangularMatrixTransposed = Device_MatrixTranspose(d_triangularMatrix, rowAndColNum, rowAndColNum);

	//Set the transposes diagonal to zero
	dim3 blockDim(32);
	dim3 gridDim(1, ceilf(rowAndColNum / (float)32));
	setDiagonalToZero<<<gridDim, blockDim>>>(d_triangularMatrixTransposed, rowAndColNum);

	//TODO: Debug, remove this. It will affect performance
	CudaCheckError();


	//Add the triangular matrices together
	float alpha = 1.0f;
	float beta = 1.0f;

	//Allocate memory for the full matrix
	hipMalloc(&d_fullMatrix, sizeof(float) * rowAndColNum * rowAndColNum);

	hipblasStatus_t hipblasStatus_t = hipblasSgeam(*cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, rowAndColNum, rowAndColNum,
			&alpha, d_triangularMatrix, rowAndColNum, &beta, d_triangularMatrixTransposed, rowAndColNum, d_fullMatrix, rowAndColNum);

	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_FullSymmetricMatrix: hipblasSgeam call failed\n");
		exit(1);
	}

	//Free the transposed matrix
	hipFree(d_triangularMatrixTransposed);

	//return the result
	return d_fullMatrix;
}




void Device_EigenvalueSolver(RFIMMemoryStruct* RFIMStruct)
{


	hipsolverStatus_t cusolverStatus;



	cusolverStatus = hipsolverDnSgesvd(*RFIMStruct->cusolverHandle, 'A', 'A', RFIMStruct->h_valuesPerSample, RFIMStruct->h_valuesPerSample,
			RFIMStruct->d_fullSymmetricCovarianceMatrix, RFIMStruct->h_valuesPerSample, RFIMStruct->d_S,  RFIMStruct->d_U, RFIMStruct->h_valuesPerSample, RFIMStruct->d_VT, RFIMStruct->h_valuesPerSample,
			RFIMStruct->d_eigWorkingSpace, RFIMStruct->h_eigWorkingSpaceLength, NULL, RFIMStruct->d_devInfo);


	/*
	int* h_devInfo = (int*)malloc(sizeof(int));
	hipMemcpy(h_devInfo, RFIMStruct->d_devInfo, sizeof(int), hipMemcpyDeviceToHost);

	if(*h_devInfo != 0)
	{
		fprintf(stderr, "Device_EigenvalueSolver: Error with the %dth parameter\n", *h_devInfo);
		//exit(1);
	}

	free(h_devInfo);



	if(cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
	{

		if(cusolverStatus == HIPSOLVER_STATUS_NOT_INITIALIZED)
			printf("1\n");
		if(cusolverStatus == HIPSOLVER_STATUS_INVALID_VALUE)
			printf("2\n");
		if(cusolverStatus == HIPSOLVER_STATUS_ARCH_MISMATCH)
			printf("3\n");
		if(cusolverStatus == HIPSOLVER_STATUS_INTERNAL_ERROR)
			printf("4\n");



		fprintf(stderr, "Device_EigenvalueSolver: Error solving eigenvalues\n");
		exit(1);
	}

}



//Eigenvector reduction and signal projection/filtering
//All matrices are column-major

//h_eigenVectorDimensionsToReduce is the number of eigenvectors to remove from the eigenvector matrix, for now it's 2

//Os = Original signal matrix
//A column-major matrix containing the signal
//It has dimensionality: h_valuesPerSample * h_numberOfSamples, which will probably be 26 x 1024?

//Er = Reduced Eigenvector matrix.
//The eigenvectors of the fully symmetrical covariance matrix, with some of the eigenvectors removed.
//It has dimensions: h_valuesPerSample x (h_valuesPerSample - h_eigenVectorDimensionsToReduce), probably 26 x 24?

//Ps = Projected signal matrix.
//The original data projected along the reduced eigenvector axis's
//This matrix will have dimensions: (h_valuesPerSample - h_eigenVectorDimensionsToReduce) x h_numberOfSamples, probably 24 x 1024?

//Fs = Final signal matrix
//This is the original data projected into the lower reduced eigenvector dimensionality, then back into the original dimensionality. This has the effect of flattening data along the removed dimensions. It may add correlations were there was previously none?
//But should also hopefully remove some RFI
//It will have dimensions: h_valuesPerSample * h_numberOfSamples, probably 26 x 1024?


//Equations!
// Ps = (Er Transposed) * Os
// Fs = Er * Ps      Note that the inverse of Er should just be its transpose, even if you remove some of the eigenvectors. This is because all the eigenvectors are orthogonal unit vectors (or should be anyway...)


//Steps
//1. Remove RFIMStruct->h_eigenVectorDimensionsToReduce dimensions from the eigenvector matrix (this is done via pointer arithmetic rather than actually removing the data) THIS WON'T WORK IF THE COLUMNS TO REMOVE ARE NOT ALL NEXT TO EACH OTHER!
//2. Compute the matrix Ps
//3. Compute the matrix Fs (final signal matrix)
//4. Pass on Fs, down the line? Keep it on the GPU? Copy it to the host? Write it to a file in the file system? Dunno.



void Device_EigenReductionAndFiltering(RFIMMemoryStruct* RFIMStruct, float* d_originalSignalMatrix, float* d_filteredSignal)
{

	hipblasStatus_t hipblasStatus_t;

	//Projected signal matrix
	//Ps = (Er Transposed) * Os
	float alpha = 1;
	float beta = 0;

	uint32_t reducedDimension = RFIMStruct->h_valuesPerSample - RFIMStruct->h_eigenVectorDimensionsToReduce;
	uint32_t eigenvectorPointerOffset = RFIMStruct->h_valuesPerSample * RFIMStruct->h_eigenVectorDimensionsToReduce;

	hipblasStatus_t = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			reducedDimension, RFIMStruct->h_numberOfSamples, RFIMStruct->h_valuesPerSample,
			&alpha,  RFIMStruct->d_U + eigenvectorPointerOffset, RFIMStruct->h_valuesPerSample,
			d_originalSignalMatrix, RFIMStruct->h_valuesPerSample, &beta,
			RFIMStruct->d_projectedSignalMatrix, reducedDimension);

	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_EigenReductionAndFiltering: error calculating the projected signal\n");
		exit(1);
	}


	//final signal matrix
	// Fs = Er * Ps

	hipblasStatus_t = hipblasSgemm(*RFIMStruct->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			RFIMStruct->h_valuesPerSample, RFIMStruct->h_numberOfSamples, reducedDimension,
			&alpha, RFIMStruct->d_U + eigenvectorPointerOffset, RFIMStruct->h_valuesPerSample,
			RFIMStruct->d_projectedSignalMatrix, reducedDimension, &beta,
			d_filteredSignal, RFIMStruct->h_valuesPerSample);


	if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "Device_EigenReductionAndFiltering: error calculating the filtered signal\n");
		exit(1);
	}

}

*/


