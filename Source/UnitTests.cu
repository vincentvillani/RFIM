#include "hip/hip_runtime.h"
/*
 * UnitTests.cu
 *
 *  Created on: 10/03/2016
 *      Author: vincentvillani
 */


#include "../Header/UnitTests.h"

#include "../Header/CudaMacros.h"
#include "../Header/CudaUtilityFunctions.h"
#include "../Header/Kernels.h"
#include "../Header/RFIMHelperFunctions.h"
#include "../Header/UtilityFunctions.h"
#include "../Header/RFIMMemoryStruct.h"

#include <hipblas.h>

#include <assert.h>
#include <cmath>
#include <string>


//Production tests
void MeanCublasProduction();
void CovarianceCublasProduction();
void EigendecompProduction();
void FilteringProduction();
//void TransposeProduction();
//void GraphProduction();





//-------------------------------------

//Production
//-------------------------------------

void MeanCublasProduction()
{

	uint32_t valuesPerSample = 3;
	uint32_t sampleNum = 2;

	RFIMMemoryStruct* RFIMStruct = RFIMMemoryStructCreate(valuesPerSample, sampleNum, 2);


	float* h_signal = (float*)malloc(sizeof(float) * valuesPerSample * sampleNum);
	float* d_signal;
	hipMalloc(&d_signal, sizeof(float) * valuesPerSample * sampleNum);

	//Set the host signal
	for(uint32_t i = 0; i < valuesPerSample * sampleNum; ++i)
	{
		h_signal[i] = i + 1;
	}

	CudaUtility_CopySignalToDevice(h_signal, &d_signal,  sizeof(float) * valuesPerSample * sampleNum);

	//Calculate the mean matrix
	Device_CalculateMeanMatrix(RFIMStruct, d_signal);


	//Copy it back to the host
	//At this point d_upperTriangularCovarianceMatrix is the mean matrix
	float* h_meanMatrix = (float*)malloc(sizeof(float) * valuesPerSample * valuesPerSample);
	CudaUtility_CopySignalToHost(RFIMStruct->d_upperTriangularCovarianceMatrix, &h_meanMatrix, valuesPerSample * valuesPerSample * sizeof(float));

	/*
	//Print out the result
	for(uint32_t i = 0; i < valuesPerSample * valuesPerSample; ++i)
	{
		printf("final: %u: %f\n", i, h_meanMatrix[i]);
	}
	*/



	bool failed = false;

	if(fabsf(h_meanMatrix[0] - 6.25f) > 0.000001f)
		failed = true;
	if(fabsf(h_meanMatrix[1]) > 0.000001f)
		failed = true;
	if(fabsf(h_meanMatrix[2]) > 0.000001f)
		failed = true;
	if(fabsf(h_meanMatrix[3] - 8.75f) > 0.000001f)
		failed = true;
	if(fabsf(h_meanMatrix[4] - 12.25f) > 0.000001f)
		failed = true;
	if(fabsf(h_meanMatrix[5]) > 0.000001f)
		failed = true;
	if(fabsf(h_meanMatrix[6] - 11.25f) > 0.000001f)
		failed = true;
	if(fabsf(h_meanMatrix[7] - 15.75f) > 0.000001f)
		failed = true;
	if(fabsf(h_meanMatrix[8] - 20.25f) > 0.000001f)
		failed = true;


	if(failed)
	{
		fprintf(stderr, "MeanCublasProduction failed!\n");
		exit(1);
	}

	RFIMMemoryStructDestroy(RFIMStruct);

}




void CovarianceCublasProduction()
{
	uint64_t valuesPerSample = 3;
	uint64_t sampleNum = 2;

	RFIMMemoryStruct* RFIMStruct = RFIMMemoryStructCreate(valuesPerSample, sampleNum, 2);

	float* h_signal; //Column first signal (3, 2), 3 == valuesPerSample, 2 == sampleNum

	h_signal = (float*)malloc( sizeof(float) * valuesPerSample * sampleNum);


	//Set the host signal
	for(uint32_t i = 0; i < valuesPerSample * sampleNum; ++i)
	{
		h_signal[i] = i + 1;
	}

	float* d_signal;
	hipMalloc(&d_signal, sizeof(float) * valuesPerSample * sampleNum);

	CudaUtility_CopySignalToDevice(h_signal, &d_signal, sizeof(float) * valuesPerSample * sampleNum);
	free(h_signal);

	//Calculate the covariance matrix
	Device_CalculateCovarianceMatrix(RFIMStruct, d_signal);

	//Copy the data back to the device and print it


	//Test the upper tricovariance matrix
	float* h_upperTriCovarMatrix = (float*)malloc(sizeof(float) * valuesPerSample * valuesPerSample);

	CudaUtility_CopySignalToHost(RFIMStruct->d_upperTriangularCovarianceMatrix, &h_upperTriCovarMatrix, valuesPerSample * valuesPerSample * sizeof(float));

	/*
	for(int i = 0; i < valuesPerSample * valuesPerSample; ++i)
	{
		printf("Covar %d: %f\n", i, h_upperTriCovarMatrix[i]);
	}
	*/


	bool failed = false;


	if(fabsf(h_upperTriCovarMatrix[0] - 2.25f) > 0.000001f)
	{
		failed = true;
	}

	if(fabsf(h_upperTriCovarMatrix[1] - 0.0f) > 0.000001f)
	{
		failed = true;
	}

	if(fabsf(h_upperTriCovarMatrix[2] - 0.0f) > 0.000001f)
	{
		failed = true;
	}

	if(fabsf(h_upperTriCovarMatrix[3] - 2.25f) > 0.000001f)
	{
		failed = true;
	}

	if(fabsf(h_upperTriCovarMatrix[4] - 2.25f) > 0.000001f)
	{
		failed = true;
	}

	if(fabsf(h_upperTriCovarMatrix[5] - 0.0f) > 0.000001f)
	{
		failed = true;
	}

	if(fabsf(h_upperTriCovarMatrix[6] - 2.25f) > 0.000001f)
	{
		failed = true;
	}

	if(fabsf(h_upperTriCovarMatrix[7] - 2.25f) > 0.000001f)
	{
		failed = true;
	}

	if(fabsf(h_upperTriCovarMatrix[8] - 2.25f) > 0.000001f)
	{
		failed = true;
	}

	if(failed)
	{
		fprintf(stderr, "CovarianceCublasProduction Unit test failed!\n");
		exit(1);
	}



	//Test the full matrix
	float* h_fullCovarianceMatrix = (float*)malloc(sizeof(float) * valuesPerSample * valuesPerSample);
	CudaUtility_CopySignalToHost(RFIMStruct->d_fullSymmetricCovarianceMatrix, &h_fullCovarianceMatrix, sizeof(float) * valuesPerSample * valuesPerSample);


	for(int i = 0; i < valuesPerSample * valuesPerSample; ++i)
	{
		if(h_fullCovarianceMatrix[i] - 2.25f > 0.000001f)
			failed = true;


		//printf("Full covar %d: %f\n", i, h_fullCovarianceMatrix[i]);
	}


	if(failed)
	{
		fprintf(stderr, "CovarianceCublasProduction Unit test at generating the correct full covariance matrix!\n");
		exit(1);
	}


	RFIMMemoryStructDestroy(RFIMStruct);

	free(h_upperTriCovarMatrix);
	free(h_fullCovarianceMatrix);
	hipFree(d_signal);
}



void EigendecompProduction()
{
	int valuesPerSample = 2;
	int covarianceMatrixByteSize = sizeof(float) * valuesPerSample * valuesPerSample;



	RFIMMemoryStruct* RFIM = RFIMMemoryStructCreate(valuesPerSample, valuesPerSample, 2);


	//Create small full covariance matrix
	float* h_fullSymmCovarianceMatrix = (float*)malloc( covarianceMatrixByteSize );

	h_fullSymmCovarianceMatrix[0] = 5.0f;
	h_fullSymmCovarianceMatrix[1] = 2.0f;
	h_fullSymmCovarianceMatrix[2] = 2.0f;
	h_fullSymmCovarianceMatrix[3] = 5.0f;



	CudaUtility_CopySignalToDevice(h_fullSymmCovarianceMatrix, &RFIM->d_fullSymmetricCovarianceMatrix,  covarianceMatrixByteSize);

	//Compute the eigenvectors/values
	Device_EigenvalueSolver(RFIM);


	//Check to see that everything is correct
	float* h_eigenvalues = (float*)malloc(sizeof(float) * valuesPerSample);
	float* h_eigenvectorMatrix = (float*)malloc(sizeof(float) * valuesPerSample * valuesPerSample);

	CudaUtility_CopySignalToHost(RFIM->d_S, &h_eigenvalues, sizeof(float) * valuesPerSample);
	CudaUtility_CopySignalToHost(RFIM->d_U, &h_eigenvectorMatrix, sizeof(float) * valuesPerSample * valuesPerSample);

	/*
	for(int i = 0; i < valuesPerSample; ++i)
	{
		printf("Eigenvalue %d: %f\n", i, h_eigenvalues[i]);
	}

	printf("\n");

	for(int i = 0; i < valuesPerSample * valuesPerSample; ++i)
	{
		printf("Eigenvec %d: %f\n", i, h_eigenvectorMatrix[i]);
	}
	*/


	bool failed = false;

	if(h_eigenvalues[0] - 7.0f > 0.0000001f)
		failed = true;
	if(h_eigenvalues[1] - 3.0f > 0.0000001f)
		failed = true;

	if(failed)
	{
		fprintf(stderr, "EigendecompProduction Unit test: failed to correctly calculate eigenvalues\n");
		exit(1);
	}

	if(fabs(h_eigenvectorMatrix[0] + 0.707107) > 0.000001f)
		failed = true;
	if(fabs(h_eigenvectorMatrix[1] + 0.707107) > 0.000001f)
		failed = true;
	if(fabs(h_eigenvectorMatrix[2] + 0.707107) > 0.000001f)
		failed = true;
	if(fabs(h_eigenvectorMatrix[3] - 0.707107) > 0.000001f)
		failed = true;

	if(failed)
	{
		fprintf(stderr, "EigendecompProduction Unit test: failed to correctly calculate eigenvectors\n");
		exit(1);
	}

	free(h_eigenvalues);
	free(h_eigenvectorMatrix);

	RFIMMemoryStructDestroy(RFIM);

}



//Doesn't actually prove that the filter itself works, just that the math functions are working as you would expected
//By removing 0 dimensions we should get the same signal back
void FilteringProduction()
{
	int valuesPerSample = 2;
	int signalByteSize = sizeof(float) * valuesPerSample * valuesPerSample;

	//REDUCE NOTHING! This should give us back the same signal
	RFIMMemoryStruct* RFIM = RFIMMemoryStructCreate(valuesPerSample, valuesPerSample, 0);


	//Create small full covariance matrix
	float* h_signal = (float*)malloc( signalByteSize );

	h_signal[0] = 1.0f;
	h_signal[1] = 2.0f;
	h_signal[2] = 7.0f;
	h_signal[3] = -8.0f;

	//Copy signal to the device
	float* d_signal;
	hipMalloc(&d_signal, signalByteSize);
	CudaUtility_CopySignalToDevice(h_signal, &d_signal, signalByteSize);

	//Calculate the covariance matrix
	Device_CalculateCovarianceMatrix(RFIM, d_signal);

	//Calculate the eigenvectors
	Device_EigenvalueSolver(RFIM);

	//Setup the signal output
	float* d_filteredSignal;
	hipMalloc(&d_filteredSignal, signalByteSize);


	//Do the projection
	Device_EigenReductionAndFiltering(RFIM, d_signal, d_filteredSignal);


	//Copy the signal back to the host
	float* h_filteredSignal = (float*)malloc(signalByteSize);
	CudaUtility_CopySignalToHost(d_filteredSignal, &h_filteredSignal, signalByteSize);

	bool failed = false;

	//Make sure we got the same signal back
	for(uint32_t i = 0; i < valuesPerSample * valuesPerSample; ++i)
	{
		//print the signal
		//printf("Orig %d: %f, filt %d: %f\n", i, h_signal[i], i, h_filteredSignal[i]);

		if(fabs(h_signal[i]) - fabs(h_filteredSignal[i]) > 0.0000001f)
		{
			failed = true;
		}
	}



	if(failed)
	{
		fprintf(stderr, "FilteringProduction: Unit test failed!\n");
		exit(1);
	}


	RFIMMemoryStructDestroy(RFIM);
	free(h_signal);
	free(h_filteredSignal);
	hipFree(d_signal);
	hipFree(d_filteredSignal);
}






void RunAllUnitTests()
{
	MeanCublasProduction();
	CovarianceCublasProduction();
	EigendecompProduction();
	FilteringProduction();

	printf("All tests passed!\n");

}

